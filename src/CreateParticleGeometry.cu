#include "hip/hip_runtime.h"
#include "CreateParticleGeometry.hpp"

void CreateParticleGeometry(particles &vd, std::vector<std::pair<probe_particles, int>> &vp_vec, Obstacle *&obstacle_ptr, Parameters &params, AuxiliarParameters &auxParams)
{

    Vcluster<> &v_cl = create_vcluster();

    // Non periodic situation grid of 5 fluid particles and 3 boundary particles
    // We need a virtual grid of 5 + 2*(3+1) particles,
    // therefore the domain is discretized with 13 grid points,
    // when we use DrawParticles::DrawBox we will draw only the particles at the grid positons strictly inside the box,
    // the () repesent the recipient box, and the || represent the fluid box, we can see how this distribution places exactly 5 fluid particles inside and 3 boundary particles
    //           D-(-o--o--o-|-x--x--x--x--x--|-o-o-o-)-D
    // D: domain, o: boundary, x: fluid, --: dp distance
    // in a periodic situation we have the following
    // .....--x--x--D-|-x--x--x--x--x--|-D--x--x--......
    // therefore we need a grid of 5 + 2 particles, and the domain is discretized with 7 grid points

    // Size of the virtual cartesian grid that defines where to place the particles
    size_t sz[DIM];

    obstacle_ptr = new EmptyObstacle(params);
    real_number dp = params.dp;
    // Initialize obstacle in scenarios where needed
    if (params.SCENARIO == CYLINDER_ARRAY)
        obstacle_ptr = new CylinderObstacle(params.LengthScale, params.ObstacleCenter, params, params.ObstacleVelocity, params.ObstacleOmega, params.rf);
    else if (params.SCENARIO == CYLINDER_LATTICE)
        obstacle_ptr = new CylinderObstacle(params.LengthScale, params.ObstacleCenter, params, params.ObstacleVelocity, params.ObstacleOmega, params.rf);
    else if (params.SCENARIO == SQUARE)
        obstacle_ptr = new RectangleObstacle(params.ObstacleCenter, params, params.ObstacleBase, params.ObstacleHeight, params.ObstacleVelocity, params.ObstacleOmega, params.rf);
    else if (params.SCENARIO == TRIANGLE)
        obstacle_ptr = new TriangleObstacle(params.ObstacleCenter, params, params.ObstacleBase, params.ObstacleHeight, params.ObstacleVelocity, params.ObstacleOmega, params.rf);
    else if (params.SCENARIO == TRIANGLE_TEST)
        obstacle_ptr = new TriangleTestObstacle(params.ObstacleCenter, params, params.ObstacleBase, params.ObstacleHeight, params.ObstacleVelocity, params.ObstacleOmega, params.rf);
    else if (params.SCENARIO == TRIANGLE_EQUILATERAL)
        obstacle_ptr = new TriangleEqui(params.ObstacleCenter, params, params.ObstacleBase, params.ObstacleVelocity, params.ObstacleOmega, params.rf);
    else if (params.SCENARIO == ELLIPSE)
        obstacle_ptr = new EllipticObstacle(params.ObstacleBase, params.ObstacleHeight, params.ObstacleTilt, params.ObstacleCenter, params, params.ObstacleVelocity, params.ObstacleOmega, params.rf);
    else if (params.SCENARIO == FLOWER)
    {
        // obstacle_ptr = new CurveObstacle(params.flowerA, params.flowerB, params.flowerK, params.flowerM, params.ObstacleCenter, params, params.ObstacleVelocity, params.ObstacleOmega, params.rf);
        obstacle_ptr = new EpiCycloid(params.flowerA, params.flowerK, params.ObstacleCenter, params, params.ObstacleVelocity, params.ObstacleOmega, params.rf);
        // obstacle_ptr = new HipoCycloid(params.flowerA, params.flowerK, params.ObstacleCenter, params, params.ObstacleVelocity, params.ObstacleOmega, params.rf);
    }
    else if (params.SCENARIO == CUSTOM)
    {
        if (params.CustomObstacle == CYLINDER_LATTICE)
            obstacle_ptr = new CylinderObstacle(params.ObstacleBase, params.ObstacleCenter, params, params.ObstacleVelocity, params.ObstacleOmega, params.rf);
        else if (params.CustomObstacle == SQUARE)
            obstacle_ptr = new RectangleObstacle(params.ObstacleCenter, params, params.ObstacleBase, params.ObstacleHeight, params.ObstacleVelocity, params.ObstacleOmega, params.rf);
        else if (params.CustomObstacle == TRIANGLE)
            obstacle_ptr = new TriangleObstacle(params.ObstacleCenter, params, params.ObstacleBase, params.ObstacleHeight, params.ObstacleVelocity, params.ObstacleOmega, params.rf);
        else if (params.CustomObstacle == TRIANGLE_EQUILATERAL)
            obstacle_ptr = new TriangleEqui(params.ObstacleCenter, params, params.ObstacleBase, params.ObstacleVelocity, params.ObstacleOmega, params.rf);
        else if (params.CustomObstacle == ELLIPSE)
            obstacle_ptr = new EllipticObstacle(params.ObstacleBase, params.ObstacleHeight, params.ObstacleTilt, params.ObstacleCenter, params, params.ObstacleVelocity, params.ObstacleOmega, params.rf);
    }

    real_number refine_factor = params.rf;

    // Now define the iterator boxes
    // We define the boxes in terms of offstes with respect to the fluid box that goes from 0 to length
    real_number offset_domain_left[DIM] = {0.0};
    real_number offset_domain_right[DIM] = {0.0};
    real_number offset_recipient[DIM] = {0.0};
    real_number offset_periodic_fluid[DIM] = {0.0};
    real_number offset_periodic_recipient[DIM] = {0.0};

    for (int xyz = 0; xyz < DIM; xyz++)
    {
        if (params.bc[xyz] == NON_PERIODIC) // non periodic, fluid covered by boundary
        {
            sz[xyz] = params.Nfluid[xyz] + 2 * (params.Nboundary[xyz] + 1);
            offset_domain_left[xyz] = (0.5 + params.Nboundary[xyz]) * dp;
            offset_domain_right[xyz] = (0.5 + params.Nboundary[xyz]) * dp;

            if (params.BC_TYPE == NEW_NO_SLIP) // Nboundary should only be 0 or 1 if we are using the new bc
                offset_recipient[xyz] = 0.1 * params.Nboundary[xyz] * dp;
            else if (params.BC_TYPE == NO_SLIP)
                offset_recipient[xyz] = params.Nboundary[xyz] * dp;
        }
        else // periodic, open ended
        {
            sz[xyz] = params.Nfluid[xyz] + 1;

            offset_domain_left[xyz] = 0.0;
            offset_domain_right[xyz] = dp;
            offset_periodic_fluid[xyz] = 0.75 * dp;
            offset_periodic_recipient[xyz] = 0.85 * dp;
        }
    }

    // Define the boxes
    Box<DIM, real_number> domain({-offset_domain_left[0],
                                  -offset_domain_left[1]},
                                 {params.length[0] + offset_domain_right[0],
                                  params.length[1] + offset_domain_right[1]});

    Box<DIM, real_number> fluid_box({0.0,
                                     0.0},
                                    {params.length[0] + offset_periodic_fluid[0],
                                     params.length[1] + offset_periodic_fluid[1]});

    Box<DIM, real_number> recipient({-offset_recipient[0],
                                     -offset_recipient[1]},
                                    {params.length[0] + offset_recipient[0] + offset_periodic_recipient[0],
                                     params.length[1] + offset_recipient[1] + offset_periodic_recipient[1]});

    // Will only be used in the new bc
    // Box<DIM, real_number> recipient_hole({offset_recipient[0],
    //                                       offset_recipient[1]},
    //                                      {params.length[0] - offset_recipient[0] + offset_periodic_fluid[0],
    //                                       params.length[1] - offset_recipient[1] + offset_periodic_fluid[1]});

    for (int xyz = 0; xyz < DIM; xyz++) // correct length in periodic case
    {
        if (params.bc[xyz] == PERIODIC)
            params.length[xyz] += dp;
    }

    // extended boundary around the domain, and the processor domain
    Ghost<DIM, real_number> g(params.r_cut);

    // create particle object
    particles vd_loc(0, domain, params.bc, g, DEC_GRAN(128));
    // vd is argument passed as reference we want to fill with particles
    vd = vd_loc;

    // place probes
    if (params.PROBES_ENABLED)
    {
        // In practice this is only used in the Cylinder Lattice scenario
        // we want to place probes  in a vertical line at this locations

        Point<DIM, real_number> EndChannel = {params.length[0], 0.0};
        Point<DIM, real_number> HalfChannel = {params.length[0] / 2.0, 0.0};
        Point<DIM, real_number> VerticalOffset = {0.0, dp};
        Point<DIM, real_number> HorizontalOffset = {dp, 0.0};
        int k0 = 0;
        int kendHeight = params.Nfluid[1] + 1;

        std::vector<Point<DIM, real_number>> ProbePoints; // start points for the PlaceProbes function
        std::vector<int> ProbeComponents;                 // velocity component to measure 0 for x 1 for y
        std::vector<Point<DIM, real_number>> Offsets;
        std::vector<int> maxIters;

        if (params.SCENARIO == CYLINDER_LATTICE)
        {
            ProbePoints.push_back(HalfChannel);
            ProbePoints.push_back(EndChannel);

            ProbeComponents.push_back(0); // measure x velocity
            ProbeComponents.push_back(0); // measure x velocity

            Offsets.push_back(VerticalOffset);
            Offsets.push_back(VerticalOffset);

            maxIters.push_back(kendHeight);
            maxIters.push_back(kendHeight);
        }

        for (unsigned int k = 0; k < ProbePoints.size(); k++) // for each probe in the list
        {
            // create probe object
            Ghost<DIM, real_number> gp(0);
            size_t bc_p[DIM] = {NON_PERIODIC, NON_PERIODIC};
            probe_particles vp_loc(0, domain, bc_p, gp, DEC_GRAN(512));
            if (ProbeComponents[k] == 0)
            {
                openfpm::vector<std::string> names_p = {"vx"};
                vp_loc.setPropNames(names_p);
            }
            else if (ProbeComponents[k] == 1)
            {
                openfpm::vector<std::string> names_p = {"vy"};
                vp_loc.setPropNames(names_p);
            }

            if (v_cl.getProcessUnitID() == 0)
            {
                PlaceProbes(vp_loc, k0, maxIters[k], ProbePoints[k], Offsets[k], obstacle_ptr);
            }
            std::pair<probe_particles, int> tmp = std::make_pair(vp_loc, ProbeComponents[k]);
            vp_vec.push_back(tmp);
            auxParams.probe_filenames.push_back("probes_" + std::to_string(k) + "_" + auxParams.filename);
        }
    }

    // For the new bc we place the wall/obstacle particles manually
    // Add the obstacle/walls as marker particles only on processor 0
    if (params.BC_TYPE == NEW_NO_SLIP && v_cl.getProcessUnitID() == 0)
    {
        // Add obstacle
        obstacle_ptr->AddObstacle(vd);

        // Add walls
        if (params.bc[0] == PERIODIC && params.bc[1] == NON_PERIODIC) // Channel like scenario
        {
            real_number dx_wall = dp / refine_factor;
            int Nwall = ceil(params.length[0] / dx_wall);
            dx_wall = params.length[0] / Nwall;
            Point<DIM, real_number> X_Offset = {dx_wall, 0.0};

            Point<DIM, real_number> LL_corner = {0.0, 0.0};
            Point<DIM, real_number> UL_corner = {0.0, params.length[1]};
            // Top And Bottom Walls

            if (params.SCENARIO == CUSTOM)
            {
                AddFlatWallNewBC(vd, 0, Nwall, LL_corner, X_Offset, dx_wall, {0.0, 0.0}, params.vw_bottom, params, FREE_SLIP_BOUNDARY, 0.0);
                AddFlatWallNewBC(vd, 0, Nwall, UL_corner, X_Offset, dx_wall, {0.0, 0.0}, params.vw_top, params, FREE_SLIP_BOUNDARY, 0.0);
            }
            else
            {
                AddFlatWallNewBC(vd, 0, Nwall, LL_corner, X_Offset, dx_wall, {0.0, 0.0}, params.vw_bottom, params, BOUNDARY, 0.0);
                AddFlatWallNewBC(vd, 0, Nwall, UL_corner, X_Offset, dx_wall, {0.0, 0.0}, params.vw_top, params, BOUNDARY, 0.0);
            }
        }
        else if (params.bc[0] == NON_PERIODIC && params.bc[1] == NON_PERIODIC) // Box like scenario
        {
            real_number dx_wall_x = dp / refine_factor;
            int Nwall_x = ceil(params.length[0] / dx_wall_x);
            dx_wall_x = params.length[0] / Nwall_x;
            Point<DIM, real_number> X_Offset = {dx_wall_x, 0.0};

            real_number dx_wall_y = dp / refine_factor;
            int Nwall_y = ceil(params.length[1] / dx_wall_y);
            dx_wall_y = params.length[1] / Nwall_y;
            Point<DIM, real_number> Y_Offset = {0.0, dx_wall_y};

            Point<DIM, real_number> LL_corner = {0.0, 0.0};
            Point<DIM, real_number> LR_corner = {params.length[0], 0.0};

            Point<DIM, real_number> UL_corner = {0.0, params.length[1]};

            // Top And Bottom Walls
            AddFlatWallNewBC(vd, 0, Nwall_x + 1, LL_corner, X_Offset, dx_wall_x, {0.0, 0.0}, params.vw_bottom, params, BOUNDARY, 0.0);
            AddFlatWallNewBC(vd, 0, Nwall_x + 1, UL_corner, X_Offset, dx_wall_x, {0.0, 0.0}, params.vw_top, params, BOUNDARY, 0.0);

            // Left And Right Walls
            AddFlatWallNewBC(vd, 1, Nwall_y, LL_corner, Y_Offset, dx_wall_y, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, 0.0);
            AddFlatWallNewBC(vd, 1, Nwall_y, LR_corner, Y_Offset, dx_wall_y, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, 0.0);
        }
    }

    // Now place fluid particles
    // For the old bc, some of the fluid positions will be inside the obstacle
    // if that is true we need to set them as boundary particles
    // For the new bc, we will not add particles that are inside the obstacle, since we already added them

    // return an iterator to the fluid particles to add to vd
    auto fluid_it = DrawParticles::DrawBox(vd, sz, domain, fluid_box);

    // for each particle inside the fluid box ...
    while (fluid_it.isNext())
    {

        Point<DIM, real_number> iterator_position = fluid_it.get();

        if ((*obstacle_ptr).isInside(iterator_position)) // if inside the obstacle region
        {
            if (params.BC_TYPE == NO_SLIP) // add particle but set it as boundary
            {
                // ... add a particle ...
                vd.add();
                vd.template getLastProp<vd0_type>() = OBSTACLE;
                vd.template getLastProp<vd10_omega>() = (*obstacle_ptr).AngularVelocity_;
                for (int xyz = 0; xyz < DIM; xyz++)
                {
                    vd.template getLastProp<vd4_velocity>()[xyz] = ((*obstacle_ptr).LinearVelocity_).get(xyz);
                    vd.template getLastProp<vd7_force_t>()[xyz] = ((*obstacle_ptr).Centre_).get(xyz);
                }
            }
            else if (params.BC_TYPE == NEW_NO_SLIP) // not add particle because already added
            {
                ++fluid_it;
                continue;
            }
        }
        else // if not inside obstacle at all just add fluid particles
        {
            // ... add a particle ...
            vd.add();
            vd.template getLastProp<vd0_type>() = FLUID;
            vd.template getLastProp<vd10_omega>() = 0.0;
            for (int xyz = 0; xyz < DIM; xyz++)
            {

                // if (params.SCENARIO == CUSTOM)
                // {
                //     vd.template getLastProp<vd4_velocity>()[xyz] = params.Vinflow[xyz];
                // }
                // else
                // {
                vd.template getLastProp<vd4_velocity>()[xyz] = 0.0;
                // }
                // else
                // {
                // vd.template getLastProp<vd4_velocity>()[xyz] = 0.0;
                // }

                vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
            }
        }

        // Set properties common to fluid and obstacle particles
        vd.template getLastProp<vd1_rho>() = params.rho0;
        vd.template getLastProp<vd2_pressure>() = 0.0;
        vd.template getLastProp<vd3_drho>() = 0.0;

        for (int xyz = 0; xyz < DIM; xyz++)
        {
            //  random displacement ( useful for debugging )
            // double random_number = static_cast<double>(std::rand()) / RAND_MAX * 2.0 - 1.0;

            vd.getLastPos()[xyz] = iterator_position.get(xyz); //+ 0.1 * dp * random_number;
            vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
            vd.template getLastProp<vd8_normal>()[xyz] = 0.0;
            vd.template getLastProp<vd6_force>()[xyz] = 0.0;
        }

        // fluid particles carry dp in volume[0], it is useful to read it in postprocessing
        // this array is unused for fluid particles anyway
        vd.template getLastProp<vd9_volume>()[0] = dp;
        vd.template getLastProp<vd9_volume>()[1] = 0.0;
        vd.template getLastProp<vd9_volume>()[2] = 0.0;

        vd.template getLastProp<vd11_vorticity>() = 0.0;
        vd.template getLastProp<vd12_vel_red>() = 0.0;
        vd.template getLastProp<vd13_force_red_x>() = 0.0;
        vd.template getLastProp<vd14_force_red_y>() = 0.0;

        // next fluid particle
        ++fluid_it;
    }

    // Now place solid walls using iterators (only for OLD BC)

    if (params.BC_TYPE == NO_SLIP)
    {

        openfpm::vector<Box<DIM, real_number>> holes;
        holes.add(fluid_box);
        Box<DIM, real_number> hole_box = holes.get(0);
        auto bound_box = DrawParticles::DrawSkin(vd, sz, domain, holes, recipient);

        if (params.bc[0] != PERIODIC || params.bc[1] != PERIODIC) // no walls in all periodic scenario
        {
            while (bound_box.isNext())
            {
                Point<DIM, real_number> position = bound_box.get();

                // periodic bc, with no boundary particles in y direction has a bug, it puts 3 extra particles outside in the y direction
                // When running on multiple cores, with this we check if particle is outside the recipient box
                // Another bug places boundary particles in the correct plane, but inside the fluid box;
                // if (bc[0] == PERIODIC && position.get(0) > dp / 2.0 && position.get(0) < length[0] - dp / 2.0)
                // {
                // 	++bound_box;
                // 	continue;
                // }

                if (!recipient.isInside((position)))
                {
                    ++bound_box;
                    continue;
                }
                if (hole_box.isInside(position))
                {
                    ++bound_box;
                    continue;
                }

                vd.add();

                vd.template getLastProp<vd0_type>() = BOUNDARY;
                vd.template getLastProp<vd1_rho>() = params.rho0;
                vd.template getLastProp<vd2_pressure>() = 0.0;
                vd.template getLastProp<vd3_drho>() = 0.0;

                for (int xyz = 0; xyz < DIM; xyz++)
                {
                    vd.getLastPos()[xyz] = bound_box.get().get(xyz);
                    vd.template getLastProp<vd6_force>()[xyz] = 0.0;
                    vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
                    vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
                    vd.template getLastProp<vd8_normal>()[xyz] = 0.0;
                    if (position.get(1) < dp / 4.0) // bottom wall
                    {
                        vd.template getLastProp<vd4_velocity>()[xyz] = params.vw_bottom[xyz];
                    }
                    else if (position.get(1) > params.length[1] - dp / 4.0) // top wall
                    {
                        vd.template getLastProp<vd4_velocity>()[xyz] = params.vw_top[xyz];
                    }
                }

                vd.template getLastProp<vd9_volume>()[0] = dp;
                vd.template getLastProp<vd9_volume>()[1] = 0.0;
                vd.template getLastProp<vd9_volume>()[2] = 0.0;
                vd.template getLastProp<vd11_vorticity>() = 0.0;
                vd.template getLastProp<vd12_vel_red>() = 0.0;
                vd.template getLastProp<vd13_force_red_x>() = 0.0;
                vd.template getLastProp<vd14_force_red_y>() = 0.0;

                ++bound_box;
            }
        }
    }
}

void CreateParticleGeometryPoiseuilleTank(particles &vd, Parameters &params, AuxiliarParameters &auxParams)
{

    Vcluster<> &v_cl = create_vcluster();

    // Non periodic situation grid of 5 fluid particles and 3 boundary particles
    // We need a virtual grid of 5 + 2*(3+1) particles,
    // therefore the domain is discretized with 13 grid points,
    // when we use DrawParticles::DrawBox we will draw only the particles at the grid positons strictly inside the box,
    // the () repesent the recipient box, and the || represent the fluid box, we can see how this distribution places exactly 5 fluid particles inside and 3 boundary particles
    //           D-(-o--o--o-|-x--x--x--x--x--|-o-o-o-)-D
    // D: domain, o: boundary, x: fluid, --: dp distance
    // in a periodic situation we have the following
    // .....--x--x--D-|-x--x--x--x--x--|-D--x--x--......
    // therefore we need a grid of 5 + 2 particles, and the domain is discretized with 7 grid points

    // Size of the virtual cartesian grid that defines where to place the particles
    size_t sz[DIM];

    real_number dp = params.dp;

    real_number refine_factor = params.rf;

    // Now define the iterator boxes
    // We define the boxes in terms of offstes with respect to the fluid box that goes from 0 to length
    real_number offset_domain_left[DIM] = {0.0};
    real_number offset_domain_right[DIM] = {0.0};
    real_number offset_recipient[DIM] = {0.0};
    real_number offset_periodic_fluid[DIM] = {0.0};
    real_number offset_periodic_recipient[DIM] = {0.0};

    for (int xyz = 0; xyz < DIM; xyz++)
    {
        if (params.bc[xyz] == NON_PERIODIC) // non periodic, fluid covered by boundary
        {
            sz[xyz] = params.Nfluid[xyz] + 2 * (params.Nboundary[xyz] + 1);
            offset_domain_left[xyz] = (0.5 + params.Nboundary[xyz]) * dp;
            offset_domain_right[xyz] = (0.5 + params.Nboundary[xyz]) * dp;

            if (params.BC_TYPE == NEW_NO_SLIP) // Nboundary should only be 0 or 1 if we are using the new bc
                offset_recipient[xyz] = 0.1 * params.Nboundary[xyz] * dp;
            else if (params.BC_TYPE == NO_SLIP)
                offset_recipient[xyz] = params.Nboundary[xyz] * dp;
        }
        else // periodic, open ended
        {
            sz[xyz] = params.Nfluid[xyz] + 1;

            offset_domain_left[xyz] = 0.0;
            offset_domain_right[xyz] = dp;
            offset_periodic_fluid[xyz] = 0.75 * dp;
            offset_periodic_recipient[xyz] = 0.85 * dp;
        }
    }

    // auxiliary tank length and heigth
    real_number tank_length = params.length[0] - 10.0 * dp;
    real_number tank_height = 10.0 * dp;

    sz[1] += 20;

    Box<DIM, real_number> second_fluid_box({5.0 * dp,
                                            params.length[1] + tank_height},
                                           {params.length[0] - 5.0 * dp,
                                            params.length[1] + 2.0 * tank_height});

    // Define the boxes
    Box<DIM, real_number> domain({-offset_domain_left[0],
                                  -offset_domain_left[1]},
                                 {params.length[0] + offset_domain_right[0],
                                  params.length[1] + offset_domain_right[1] + 2.0 * tank_height});

    Box<DIM, real_number> fluid_box({0.0,
                                     0.0},
                                    {params.length[0] + offset_periodic_fluid[0],
                                     params.length[1] + offset_periodic_fluid[1]});

    Box<DIM, real_number> recipient({-offset_recipient[0],
                                     -offset_recipient[1]},
                                    {params.length[0] + offset_recipient[0] + offset_periodic_recipient[0],
                                     params.length[1] + offset_recipient[1] + offset_periodic_recipient[1]});

    for (int xyz = 0; xyz < DIM; xyz++) // correct length in periodic case
    {
        if (params.bc[xyz] == PERIODIC)
            params.length[xyz] += dp;
    }

    // extended boundary around the domain, and the processor domain
    Ghost<DIM, real_number> g(params.r_cut);

    // create particle object
    particles vd_loc(0, domain, params.bc, g, DEC_GRAN(128));
    // vd is argument passed as reference we want to fill with particles
    vd = vd_loc;

    // For the new bc we place the wall/obstacle particles manually
    // Add the obstacle/walls as marker particles only on processor 0
    if (params.BC_TYPE == NEW_NO_SLIP && v_cl.getProcessUnitID() == 0)
    {
        // Add walls
        if (params.bc[0] == PERIODIC && params.bc[1] == NON_PERIODIC) // Channel like scenario
        {
            real_number dx_wall = dp / refine_factor;
            int Nwall = ceil(params.length[0] / dx_wall);
            dx_wall = params.length[0] / Nwall;
            Point<DIM, real_number> X_Offset = {dx_wall, 0.0};
            Point<DIM, real_number> Y_Offset = {0.0, dx_wall};

            Point<DIM, real_number> LL_corner = {0.0, 0.0};
            Point<DIM, real_number> UL_corner = {0.0, params.length[1]};

            // Top And Bottom Walls
            AddFlatWallNewBC(vd, 0, Nwall, LL_corner, X_Offset, dx_wall, {0.0, 0.0}, params.vw_bottom, params, BOUNDARY, 0.0);
            AddFlatWallNewBC(vd, 0, Nwall, UL_corner, X_Offset, dx_wall, {0.0, 0.0}, params.vw_top, params, BOUNDARY, 0.0);

            // Add auxiliary tank on top of the channel
            Point<DIM, real_number> tank_LL_corner = {0.0 + 5.0 * dp - 0.5 * dp, params.length[1] + tank_height};
            Point<DIM, real_number> tank_UR_corner = {params.length[0] - 5.0 * dp - 0.5 * dp, params.length[1] + 2.0 * tank_height};

            int Nwall_tank = ceil(tank_length / dx_wall);
            int Nwall_tank_y = ceil(tank_height / dx_wall);
            // Top And Bottom Walls
            AddFlatWallNewBC(vd, 0, Nwall_tank + 1, tank_LL_corner, X_Offset, dx_wall, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, 0.0);
            AddFlatWallNewBC(vd, 1, Nwall_tank_y + 1, tank_LL_corner, Y_Offset, dx_wall, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, 0.0);

            AddFlatWallNewBC(vd, 0, Nwall_tank + 1, tank_UR_corner, -X_Offset, dx_wall, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, 0.0);
            AddFlatWallNewBC(vd, 1, Nwall_tank_y + 1, tank_UR_corner, -Y_Offset, dx_wall, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, 0.0);
        }
    }

    // Now place fluid particles
    // For the old bc, some of the fluid positions will be inside the obstacle
    // if that is true we need to set them as boundary particles
    // For the new bc, we will not add particles that are inside the obstacle, since we already added them

    // return an iterator to the fluid particles to add to vd
    auto fluid_it = DrawParticles::DrawBox(vd, sz, domain, fluid_box);

    // for each particle inside the fluid box ...
    while (fluid_it.isNext())
    {

        Point<DIM, real_number> iterator_position = fluid_it.get();

        // ... add a particle ...
        vd.add();
        vd.template getLastProp<vd0_type>() = FLUID;
        vd.template getLastProp<vd10_omega>() = 0.0;
        for (int xyz = 0; xyz < DIM; xyz++)
        {
            vd.template getLastProp<vd4_velocity>()[xyz] = 0.0;
            vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
        }

        // Set properties common to fluid and obstacle particles
        vd.template getLastProp<vd1_rho>() = params.rho0;
        vd.template getLastProp<vd2_pressure>() = 0.0;
        vd.template getLastProp<vd3_drho>() = 0.0;

        for (int xyz = 0; xyz < DIM; xyz++)
        {
            vd.getLastPos()[xyz] = iterator_position.get(xyz);
            vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
            vd.template getLastProp<vd8_normal>()[xyz] = 0.0;
            vd.template getLastProp<vd6_force>()[xyz] = 0.0;
        }

        // fluid particles carry dp in volume[0], it is useful to read it in postprocessing
        // this array is unused for fluid particles anyway
        vd.template getLastProp<vd9_volume>()[0] = dp;
        vd.template getLastProp<vd9_volume>()[1] = 0.0;
        vd.template getLastProp<vd9_volume>()[2] = 0.0;

        vd.template getLastProp<vd11_vorticity>() = 0.0;
        vd.template getLastProp<vd12_vel_red>() = 0.0;
        vd.template getLastProp<vd13_force_red_x>() = 0.0;
        vd.template getLastProp<vd14_force_red_y>() = 0.0;

        // next fluid particle
        ++fluid_it;
    }

    auto fluid_it2 = DrawParticles::DrawBox(vd, sz, domain, second_fluid_box);
    // for each particle inside the fluid box ...
    while (fluid_it2.isNext())
    {

        Point<DIM, real_number> iterator_position = fluid_it2.get();

        // ... add a particle ...
        vd.add();
        vd.template getLastProp<vd0_type>() = FLUID;
        vd.template getLastProp<vd10_omega>() = 0.0;
        for (int xyz = 0; xyz < DIM; xyz++)
        {
            vd.template getLastProp<vd4_velocity>()[xyz] = 0.0;
            vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
        }

        // Set properties
        vd.template getLastProp<vd1_rho>() = params.rho0;
        vd.template getLastProp<vd2_pressure>() = 0.0;
        vd.template getLastProp<vd3_drho>() = 0.0;

        for (int xyz = 0; xyz < DIM; xyz++)
        {
            vd.getLastPos()[xyz] = iterator_position.get(xyz);
            vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
            vd.template getLastProp<vd8_normal>()[xyz] = 0.0;
        }

        vd.template getLastProp<vd9_volume>()[0] = dp;
        vd.template getLastProp<vd9_volume>()[1] = 0.0;
        vd.template getLastProp<vd9_volume>()[2] = 0.0;

        // next fluid particle
        ++fluid_it2;
    }

    // Create a Cell list object for CPU calculations
    auto NN = vd.getCellList(params.r_cut);
    vd.updateCellList(NN);

    CalcFluidVec(vd, NN, params);
    vd.ghost_get<vd0_type, vd1_rho, vd2_pressure, vd4_velocity, vd5_velocity_t, vd6_force, vd7_force_t, vd8_normal, vd9_volume, vd10_omega, vd11_vorticity>();

    CalcNormalVec(vd, NN, params);
    vd.ghost_get<vd0_type, vd1_rho, vd2_pressure, vd4_velocity, vd5_velocity_t, vd6_force, vd7_force_t, vd8_normal, vd9_volume, vd10_omega, vd11_vorticity>();

    CalcCurvature(vd, NN, params);
    vd.ghost_get<vd0_type, vd1_rho, vd2_pressure, vd4_velocity, vd5_velocity_t, vd6_force, vd7_force_t, vd8_normal, vd9_volume, vd10_omega, vd11_vorticity>();

    CalcVolume(vd, params.dp);
    vd.ghost_get<vd0_type, vd1_rho, vd2_pressure, vd4_velocity, vd5_velocity_t, vd6_force, vd7_force_t, vd8_normal, vd9_volume, vd10_omega, vd11_vorticity>();

    vd.write_frame("frametest", 0, 0.0, VTK_WRITER);
    // now move aux tank beside the dam break
    real_number ythreshold = params.length[1] + 0.5 * tank_height;
    auto it = vd.getDomainIterator();
    while (it.isNext())
    {
        auto akey = it.get();

        Point<DIM, real_number> position = vd.getPos(akey);
        if (position.get(1) > ythreshold)
        {
            vd.getPos(akey)[1] -= (tank_height - 0.5 * dp);
        }

        ++it;
    }
}

void CreateParticleGeometryTaylorCouette(particles &vd, std::vector<std::pair<probe_particles, int>> &vp_vec, Obstacle *&obstacle_ptr, Parameters params, AuxiliarParameters &auxParams)
{
    Vcluster<> &v_cl = create_vcluster();

    // Size of the virtual cartesian grid that defines where to place the particles
    size_t sz[DIM];

    real_number dp = params.dp;
    // We define the boxes in terms of offstes with respect to the fluid box that goes from 0 to length
    real_number offset_domain_left[DIM] = {0.0};
    real_number offset_domain_right[DIM] = {0.0};

    // non periodic situation grid of 5 fluid particles and 3 boundary particles
    // We need a virtual grid of 5 + 2*(3+1) particles,
    // therefore the domain is discretized with 13 grid points,
    // when we use DrawParticles::DrawBox we will draw only the particles at the grid positons strictly inside the box,
    // the () repesent the recipient box, and the || represent the fluid box, we can see how this distribution places exactly 5 fluid particles inside and 3 boundary particles
    //           D-(-o--o--o-|-x--x--x--x--x--|-o-o-o-)-D
    // D: domain, o: boundary, x: fluid, --: dp distance
    // in a periodic situation we have the following
    // .....--x--x--D-|-x--x--x--x--x--|-D--x--x--......
    // therefore we need a grid of 5 + 2 particles, and the domain is discretized with 7 grid points

    real_number Rin = params.Rin;
    real_number Rout = params.Rout;
    real_number Win = params.Win;
    real_number Wout = params.Wout;

    real_number a_tc = -((Rout * Rout * Rin * Rin) / (Rout * Rout - Rin * Rin)) * (Wout - Win);
    real_number b_tc = (Wout * Rout * Rout - Win * Rin * Rin) / (Rout * Rout - Rin * Rin);

    size_t Nbound = (params.BC_TYPE == NEW_NO_SLIP) ? 1 : 3;

    for (int dim = 0; dim < DIM; dim++)
    {
        params.length[dim] = dp * params.Nfluid[dim];
        sz[dim] = params.Nfluid[dim] + 2 * (Nbound + 1);
        offset_domain_left[dim] = (0.5 + Nbound) * dp;
        offset_domain_right[dim] = (0.5 + Nbound) * dp;
    }

    // Define the boxes
    Box<DIM, real_number> domain({-params.length[0] / 2.0 - offset_domain_left[0],
                                  -params.length[1] / 2.0 - offset_domain_left[1]},
                                 {params.length[0] / 2.0 + offset_domain_right[0],
                                  params.length[1] / 2.0 + offset_domain_right[1]});
    Box<DIM, real_number> fluid_box({-params.length[0] / 2.0,
                                     -params.length[1] / 2.0},
                                    {params.length[0] / 2.0,
                                     params.length[1] / 2.0});

    // extended boundary around the domain, and the processor domain
    Ghost<DIM, real_number> g(params.r_cut);

    // create particle object
    particles vd_loc(0, domain, params.bc, g, DEC_GRAN(128));
    vd = vd_loc;

    // Write constants on file
    real_number rf = params.rf;

    // Set cylindrical object parameters
    Point<DIM, real_number> CylinderCentre = {0.0, 0.0};

    obstacle_ptr = new EmptyObstacle(params);

    const Point<DIM, real_number> vel = {0.0, 0.0};

    CylinderObstacle *obstacle_ptr_out = new CylinderObstacle(Rout, CylinderCentre, params, vel, Wout, rf);
    CylinderObstacle *obstacle_ptr_in = new CylinderObstacle(Rin, CylinderCentre, params, vel, Win, rf);

    CylinderObstacle *obstacle_ptr_out_aux = new CylinderObstacle(Rout + 3.0 * dp, CylinderCentre, params, vel, Wout, rf);
    CylinderObstacle *obstacle_ptr_in_aux = new CylinderObstacle(Rin - 3.0 * dp, CylinderCentre, params, vel, Win, rf);

    // Add the obstacle as marker particles only on processor 0
    if (params.BC_TYPE == NEW_NO_SLIP && v_cl.getProcessUnitID() == 0)
    {
        obstacle_ptr_out->AddObstacle(vd);
        obstacle_ptr_in->AddObstacle(vd);
    }

    Box<DIM, real_number> fluid_box_aux({-Rout - 3.5 * dp,
                                         -Rout - 3.5 * dp},
                                        {Rout + 3.5 * dp,
                                         Rout + 3.5 * dp});

    // Outer Cylinder boundary particles
    auto out_it = DrawParticles::DrawBox(vd, sz, domain, fluid_box_aux);
    if (params.BC_TYPE == NO_SLIP)
    {
        while (out_it.isNext())
        {

            Point<DIM, real_number> iterator_position = out_it.get();
            if (!(*obstacle_ptr_out).isInside_minEps(iterator_position) && (*obstacle_ptr_out_aux).isInside(iterator_position)) // if outside the outer cylinder and inside outer cylinder aux
            {
                if (params.BC_TYPE == NO_SLIP)
                {
                    vd.add();
                    // Set properties
                    vd.template getLastProp<vd0_type>() = OBSTACLE;
                    vd.template getLastProp<vd10_omega>() = (*obstacle_ptr_out).AngularVelocity_;
                    for (int xyz = 0; xyz < DIM; xyz++)
                    {
                        vd.template getLastProp<vd4_velocity>()[xyz] = ((*obstacle_ptr_out).LinearVelocity_).get(xyz);
                        vd.template getLastProp<vd7_force_t>()[xyz] = ((*obstacle_ptr_out).Centre_).get(xyz);
                    }
                    vd.template getLastProp<vd1_rho>() = params.rho0;
                    vd.template getLastProp<vd2_pressure>() = 0.0;
                    vd.template getLastProp<vd3_drho>() = 0.0;

                    for (int xyz = 0; xyz < DIM; xyz++)
                    {
                        vd.getLastPos()[xyz] = iterator_position.get(xyz);
                        vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
                        vd.template getLastProp<vd8_normal>()[xyz] = 0.0;
                    }

                    vd.template getLastProp<vd9_volume>()[0] = dp;

                    // next fluid particle
                    ++out_it;
                    continue;
                }
                else
                {
                    ++out_it;
                    continue;
                }
            }
            else // skip fluid particle
            {
                ++out_it;
                continue;
            }
        }
    }

    // Inner Cylinder boundary particles and fluid particles
    auto fluid_it = DrawParticles::DrawBox(vd, sz, domain, fluid_box);
    while (fluid_it.isNext())
    {

        Point<DIM, real_number> iterator_position = fluid_it.get();
        if ((*obstacle_ptr_out).isInside_minEps(iterator_position)) // if inside the outer cylinder
        {
            if ((*obstacle_ptr_in).isInside(iterator_position)) // if inside the inner cylinder region
            {
                if (!(*obstacle_ptr_in_aux).isInside_minEps(iterator_position))
                {
                    if (params.BC_TYPE == NO_SLIP) // add particle but set it as boundary
                    {
                        // ... add a particle ...
                        vd.add();
                        vd.template getLastProp<vd0_type>() = OBSTACLE;
                        vd.template getLastProp<vd10_omega>() = (*obstacle_ptr_in).AngularVelocity_;
                        for (int xyz = 0; xyz < DIM; xyz++)
                        {
                            vd.template getLastProp<vd4_velocity>()[xyz] = ((*obstacle_ptr_in).LinearVelocity_).get(xyz);
                            vd.template getLastProp<vd7_force_t>()[xyz] = ((*obstacle_ptr_in).Centre_).get(xyz);
                        }
                    }
                    else if (params.BC_TYPE == NEW_NO_SLIP) // not add particle because already added
                    {
                        ++fluid_it;
                        continue;
                    }
                }
                else
                {
                    ++fluid_it;
                    continue;
                }
            }
            else // if no cylinder at all just add fluid particles
            {
                // ... add a particle ...
                vd.add();
                vd.template getLastProp<vd0_type>() = FLUID;
                vd.template getLastProp<vd10_omega>() = 0.0;

                // real_number r = iterator_position.get(0) * iterator_position.get(0) + iterator_position.get(1) * iterator_position.get(1);
                // r = sqrt(r);
                // real_number uth = a_tc / r + b_tc * r;

                // real_number ux = uth * (-iterator_position.get(1) / r);
                // real_number uy = uth * (iterator_position.get(0) / r);

                vd.template getLastProp<vd4_velocity>()[0] = 0.0; // ux;
                vd.template getLastProp<vd4_velocity>()[1] = 0.0; // uy;

                for (int xyz = 0; xyz < DIM; xyz++)
                {
                    vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
                }
            }
        }
        else // skip fluid particle
        {
            ++fluid_it;
            continue;
        }
        // Set properties
        vd.template getLastProp<vd1_rho>() = params.rho0;
        vd.template getLastProp<vd2_pressure>() = 0.0;
        vd.template getLastProp<vd3_drho>() = 0.0;

        for (int xyz = 0; xyz < DIM; xyz++)
        {
            vd.getLastPos()[xyz] = iterator_position.get(xyz);
            vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
            vd.template getLastProp<vd8_normal>()[xyz] = 0.0;
        }

        vd.template getLastProp<vd9_volume>()[0] = dp;

        // next fluid particle
        ++fluid_it;
    }
}
void CreateParticleGeometryStep(particles &vd, std::vector<std::pair<probe_particles, int>> &vp_vec, Parameters params, AuxiliarParameters &auxParams)
{

    Vcluster<> &v_cl = create_vcluster();
    Obstacle *obstacle_ptr = new EmptyObstacle(params);

    // Size of the virtual cartesian grid that defines where to place the particles
    size_t sz[DIM];

    real_number length_small[DIM];
    real_number length_big[DIM];
    // In the case of the new bc we need particles at the wall, for this we need sz_aux
    // We want to put one virtual grid point between each pair of the old ones,
    // so that the new spacing is dp/2, and we can put a fluid particle exactly at the wall
    size_t sz_aux[DIM];

    // Boundary conditions
    size_t bc[DIM];

    // Number of boundary particles in each direction
    size_t Nboundary_big[DIM];

    // Number of fluid particles in each direction
    size_t Nfluid_big[DIM];
    size_t Nfluid_small[DIM];

    // We define the boxes in terms of offstes with respect to the fluid box that goes from 0 to length
    real_number offset_domain[DIM] = {0.0};
    real_number offset_recipient[DIM] = {0.0};
    real_number offset_periodic_fluid[DIM] = {0.0};
    real_number offset_periodic_recipient[DIM] = {0.0};

    // non periodic situation grid of 5 fluid particles and 3 boundary particles
    // We need a virtual grid of 5 + 2*(3+1) particles,
    // therefore the domain is discretized with 13 grid points,
    // when we use DrawParticles::DrawBox we will draw only the particles at the grid positons strictly inside the box,
    // the () repesent the recipient box, and the || represent the fluid box, we can see how this distribution places exactly 5 fluid particles inside and 3 boundary particles
    //           D-(-o--o--o-|-x--x--x--x--x--|-o-o-o-)-D
    // D: domain, o: boundary, x: fluid, --: dp distance
    // in a periodic situation we have the following
    // .....--x--x--D-|-x--x--x--x--x--|-D--x--x--......
    // therefore we need a grid of 5 + 2 particles, and the domain is discretized with 7 grid points

    real_number StepHeight = 4.9;
    params.LengthScale = StepHeight;

    Nfluid_big[0] = 275;
    Nfluid_big[1] = 31;

    Nfluid_small[0] = 100;
    Nfluid_small[1] = 16;

    size_t Nbound = (params.BC_TYPE == NEW_NO_SLIP) ? 1 : 3;
    Nboundary_big[0] = 0;
    Nboundary_big[1] = Nbound;
    // real_number Nboundary_small_up = Nbound;
    // real_number Nboundary_small_down = Nbound + Nfluid_big[1] - Nfluid_small[1];

    bc[0] = PERIODIC;
    bc[1] = NON_PERIODIC;
    params.dp = params.LengthScale / ((real_number)Nfluid_big[1] - Nfluid_small[1]);
    real_number dp = params.dp;
    params.umax = 1.4 * 1e-1;

    params.H = params.Hconst * dp;
    // r_cut = (KERNEL == CUBIC ? 2.0 * H : 3.0 * H);
    params.r_cut = 3.0 * params.H;
    params.Kquintic = (DIM == 3) ? 1.0 / 120.0 / M_PI / params.H / params.H / params.H : 7.0 / 478.0 / M_PI / params.H / params.H;
    params.MassFluid = params.rho0 * (DIM == 3 ? dp * dp * dp : dp * dp);
    params.MassBound = params.rho0 * (DIM == 3 ? dp * dp * dp : dp * dp);
    params.cbar = params.coeff_sound * params.umax;
    params.B = params.rho0 * params.cbar * params.cbar / params.gamma;
    params.Pbackground = params.Bfactor * params.B;
    params.eta = params.nu * params.rho0;
    params.Re = params.umax * 2.0 * 5.2 / params.nu;

    params.gravity = getVectorNorm(params.gravity_vector);

    for (int dim = 0; dim < DIM; dim++)
    {
        if (bc[dim] == NON_PERIODIC) // non periodic, fluid covered by boundary
        {
            params.length[dim] = dp * (Nfluid_big[dim]);
            length_small[dim] = dp * (Nfluid_small[dim]);
            length_big[dim] = dp * (Nfluid_big[dim]);
            sz[dim] = Nfluid_big[dim] + 2 * (Nboundary_big[dim] + 1);
            offset_domain[dim] = (0.5 + Nboundary_big[dim]) * dp;

            if (Nboundary_big[dim] != 0)
                sz_aux[dim] = 2 * Nfluid_big[dim] - 1 + 2 * (2 * Nboundary_big[dim] + 1 + 1);
            else // for a direction with no boundary particles we dont need to add anything
                sz_aux[dim] = sz[dim];

            if (params.BC_TYPE == NEW_NO_SLIP) // Nboundary should only be 0 or 1 if we are using the new bc
                offset_recipient[dim] = 0.25 * Nboundary_big[dim] * dp;
            else if (params.BC_TYPE == NO_SLIP)
                offset_recipient[dim] = Nboundary_big[dim] * dp;
        }
        else // periodic, open ended
        {
            Nfluid_big[dim] -= 1;
            params.length[dim] = dp * (Nfluid_big[dim] + Nfluid_small[dim]);
            length_small[dim] = dp * (Nfluid_small[dim]);
            length_big[dim] = dp * (Nfluid_big[dim]);

            sz[dim] = (Nfluid_big[dim] + Nfluid_small[dim]) + 2;
            sz_aux[dim] = sz[dim];

            offset_domain[dim] = 0.5 * dp;
            offset_periodic_fluid[dim] = 0.75 * dp;
            offset_periodic_recipient[dim] = 0.85 * dp;
        }
    }

    // Define the boxes
    Box<DIM, real_number> domain({-offset_domain[0],
                                  -offset_domain[1]},
                                 {params.length[0] + offset_domain[0],
                                  params.length[1] + offset_domain[1]});

    Box<DIM, real_number> fluid_box_small({0.0,
                                           (Nfluid_big[1] - Nfluid_small[1]) * dp},
                                          {length_small[0],
                                           (Nfluid_big[1] - Nfluid_small[1]) * dp + length_small[1]});

    Box<DIM, real_number> fluid_box_big({length_small[0],
                                         0.0},
                                        {length_small[0] + length_big[0] + offset_periodic_fluid[0],
                                         length_big[1]});

    Box<DIM, real_number> recipient({-offset_recipient[0],
                                     -offset_recipient[1]},
                                    {params.length[0] + offset_recipient[0] + offset_periodic_recipient[0],
                                     params.length[1] + offset_recipient[1] + offset_periodic_recipient[1]});

    // Will only be used in the new bc
    Box<DIM, real_number> recipient_hole_small({offset_recipient[0],
                                                (Nfluid_big[1] - Nfluid_small[1]) * dp + offset_recipient[1]},
                                               {length_small[0] - offset_recipient[0],
                                                (Nfluid_big[1] - Nfluid_small[1]) * dp + length_small[1] - offset_recipient[1]});

    Box<DIM, real_number> recipient_hole_big({length_small[0] + offset_recipient[0],
                                              offset_recipient[1]},
                                             {length_small[0] + length_big[0] - offset_recipient[0] + offset_periodic_fluid[0],
                                              length_big[1] - offset_recipient[1]});

    Box<DIM, real_number> CornerHole{{3 * dp, -3 * dp}, {(3 + Nfluid_small[0] - 6) * dp, (Nfluid_big[1] - Nfluid_small[1] - 3) * dp}};
    Box<DIM, real_number> CornerHole_New{{dp, -1 * dp}, {(1 + Nfluid_small[0] - 2) * dp, (Nfluid_big[1] - Nfluid_small[1]) * dp - 0.5 * dp}};

    // extended boundary around the domain, and the processor domain
    Ghost<DIM, real_number> g(params.r_cut);

    // create particle object
    particles vd_loc(0, domain, bc, g, DEC_GRAN(128));
    vd = vd_loc;

    // correct the number of particles in case of periodicity, we substracted 1 before to accomodate the periodic boundary
    for (int dim = 0; dim < DIM; dim++)
    {
        if (bc[dim] == PERIODIC)
        {
            Nfluid_big[dim] += 1;
            params.length[dim] += dp;
        }
    }

    // Write constants on file
    // WriteParameters(v_cl, params);

    // we want to place probes  in a vertical line at this locations
    Point<DIM, real_number> P1 = {0.75 * Nfluid_small[0] * dp, 0.0};
    Point<DIM, real_number> P2 = {1.2 * Nfluid_small[0] * dp, 0.0};
    Point<DIM, real_number> P3 = {1.4 * Nfluid_small[0] * dp, 0.0};
    Point<DIM, real_number> P4 = {Nfluid_small[0] * dp + Nfluid_big[0] * dp * 0.7, 0.0};

    std::vector<Point<DIM, real_number>> ProbePoints = {P1, P2, P3, P4};

    Point<DIM, real_number> VerticalOffset = {0.0, dp};
    int k0 = 0;
    int kendHeight = Nfluid_big[1];

    // place probes
    if (params.PROBES_ENABLED)
    {
        for (int k = 0; k < 4; k++)
        {
            // create probe object
            Ghost<DIM, real_number> gp(0);
            size_t bc_p[DIM] = {NON_PERIODIC, NON_PERIODIC};
            probe_particles vp_loc(0, domain, bc_p, gp, DEC_GRAN(128));
            openfpm::vector<std::string> names_p({"vx"});
            vp_loc.setPropNames(names_p);

            PlaceProbes(vp_loc, k0, kendHeight, ProbePoints[k], VerticalOffset, obstacle_ptr);
            std::pair<probe_particles, int> tmp = std::make_pair(vp_loc, 0);
            vp_vec.push_back(tmp);
            auxParams.probe_filenames.push_back("probes_" + std::to_string(k) + "_" + auxParams.filename);
        }
    }

    // return an iterator to the fluid particles to add to vd
    auto fluid_it1 = DrawParticles::DrawBox(vd, sz, domain, fluid_box_big);
    auto fluid_it2 = DrawParticles::DrawBox(vd, sz, domain, fluid_box_small);

    // for each particle inside the fluid box ...
    while (fluid_it1.isNext())
    {

        Point<DIM, real_number> iterator_position = fluid_it1.get();

        // ... add a particle ...
        vd.add();
        vd.template getLastProp<vd0_type>() = FLUID;

        // Set properties
        vd.template getLastProp<vd1_rho>() = params.rho0;
        vd.template getLastProp<vd2_pressure>() = 0.0;
        vd.template getLastProp<vd3_drho>() = 0.0;

        for (int xyz = 0; xyz < DIM; xyz++)
        {
            vd.getLastPos()[xyz] = iterator_position.get(xyz);
            vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
            vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
            vd.template getLastProp<vd8_normal>()[xyz] = 0.0;
            vd.template getLastProp<vd4_velocity>()[xyz] = 0.0;
        }

        vd.template getLastProp<vd9_volume>()[0] = dp;

        // next fluid particle
        ++fluid_it1;
    }
    // for each particle inside the fluid box ...
    while (fluid_it2.isNext())
    {

        Point<DIM, real_number> iterator_position = fluid_it2.get();

        // ... add a particle ...
        vd.add();
        vd.template getLastProp<vd0_type>() = FLUID;

        // Set properties
        vd.template getLastProp<vd1_rho>() = params.rho0;
        vd.template getLastProp<vd2_pressure>() = 0.0;
        vd.template getLastProp<vd3_drho>() = 0.0;

        for (int xyz = 0; xyz < DIM; xyz++)
        {
            vd.getLastPos()[xyz] = iterator_position.get(xyz);
            vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
            vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
            vd.template getLastProp<vd8_normal>()[xyz] = 0.0;
            vd.template getLastProp<vd4_velocity>()[xyz] = 0.0;
        }

        vd.template getLastProp<vd9_volume>()[0] = dp;

        // next fluid particle
        ++fluid_it2;
    }

    // Now place solid walls
    openfpm::vector<Box<DIM, real_number>> holes;

    if (params.BC_TYPE == NEW_NO_SLIP)
    {
        holes.add(recipient_hole_small);
        holes.add(recipient_hole_big);
        holes.add(CornerHole_New);
        sz[0] = sz_aux[0];
        sz[1] = sz_aux[1];
    }
    else if (params.BC_TYPE == NO_SLIP)
    {
        holes.add(fluid_box_big);
        holes.add(fluid_box_small);
        holes.add(CornerHole);
    }
    Box<DIM, real_number> hole_get0 = holes.get(0);
    Box<DIM, real_number> hole_get1 = holes.get(1);
    auto bound_box = DrawParticles::DrawSkin(vd, sz, domain, holes, recipient);

    if (bc[0] != PERIODIC || bc[1] != PERIODIC) // no walls in all periodic scenario
    {
        while (bound_box.isNext())
        {
            Point<DIM, real_number> position = bound_box.get();

            // periodic bc, with no boundary particles in y direction has a bug, it puts 3 extra particles outside in the y direction
            // When running on multiple cores, with this we check if particle is outside the recipient box
            // Another bug places boundary particles in the correct plane, but inside the fluid box;
            // the first bug seems to be fixed
            // if (!recipient.isInside((position)))
            // {
            // 	++bound_box;
            // 	continue;
            // }
            // if (bc[0] == PERIODIC && position.get(0) > dp / 2.0 && position.get(0) < length[0] - dp / 2.0)
            // {
            // 	++bound_box;
            // 	continue;
            // }

            if (hole_get0.isInside(position) || hole_get1.isInside(position))
            {
                ++bound_box;
                continue;
            }

            if (params.BC_TYPE == NEW_NO_SLIP)
            {
                // Check if x and y coordinates are multiples of dp, keep multiples, discard the rest
                // real_number remx = fmod(position.get(0), dp);
                real_number remy = fmod(position.get(1), dp);
                real_number tol = 0.5 * dp * 10e-2;

                // if (remx > tol && remx < dp - tol)
                // {
                // 	++bound_box;
                // 	continue;
                // }
                if (remy > tol && remy < dp - tol)
                {
                    ++bound_box;
                    continue;
                }
            }
            vd.add();

            vd.template getLastProp<vd0_type>() = BOUNDARY;
            vd.template getLastProp<vd1_rho>() = params.rho0;
            vd.template getLastProp<vd2_pressure>() = 0.0;
            vd.template getLastProp<vd3_drho>() = 0.0;

            for (int xyz = 0; xyz < DIM; xyz++)
            {
                vd.getLastPos()[xyz] = bound_box.get().get(xyz);
                vd.template getLastProp<vd6_force>()[xyz] = 0.0;
                vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
                vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
                vd.template getLastProp<vd8_normal>()[xyz] = 0.0;
                if (position.get(1) < dp / 4.0) // bottom wall
                {
                    vd.template getLastProp<vd4_velocity>()[xyz] = params.vw_bottom[xyz];
                }
                else if (position.get(1) > params.length[1] - dp / 4.0) // top wall
                {
                    vd.template getLastProp<vd4_velocity>()[xyz] = params.vw_top[xyz];
                }
            }

            vd.template getLastProp<vd9_volume>()[0] = dp;

            ++bound_box;
        }
    }
}

void CreateParticleGeometryDamBreak(particles &vd, std::vector<std::pair<probe_particles, int>> &vp_vec, Parameters &params, AuxiliarParameters &auxParams)
{

    Vcluster<> &v_cl = create_vcluster();

    // Non periodic situation grid of 5 fluid particles and 3 boundary particles
    // We need a virtual grid of 5 + 2*(3+1) particles,
    // therefore the domain is discretized with 13 grid points,
    // when we use DrawParticles::DrawBox we will draw only the particles at the grid positons strictly inside the box,
    // the () repesent the recipient box, and the || represent the fluid box, we can see how this distribution places exactly 5 fluid particles inside and 3 boundary particles
    //           D-(-o--o--o-|-x--x--x--x--x--|-o-o-o-)-D
    // D: domain, o: boundary, x: fluid, --: dp distance
    // in a periodic situation we have the following
    // .....--x--x--D-|-x--x--x--x--x--|-D--x--x--......
    // therefore we need a grid of 5 + 2 particles, and the domain is discretized with 7 grid points

    // Size of the virtual cartesian grid that defines where to place the particles
    size_t sz[DIM];

    real_number dp = params.dp;
    real_number refine_factor = params.rf;

    // Now define the iterator boxes
    // We define the boxes in terms of offstes with respect to the fluid box that goes from 0 to length
    real_number offset_domain_left[DIM] = {0.0};
    real_number offset_domain_right[DIM] = {0.0};
    real_number offset_recipient[DIM] = {0.0};
    real_number offset_periodic_fluid[DIM] = {0.0};
    real_number offset_periodic_recipient[DIM] = {0.0};

    for (int xyz = 0; xyz < DIM; xyz++)
    {
        if (params.bc[xyz] == NON_PERIODIC) // non periodic, fluid covered by boundary
        {
            sz[xyz] = params.Nfluid[xyz] + 2 * (params.Nboundary[xyz] + 1);
            offset_domain_left[xyz] = (0.5 + params.Nboundary[xyz]) * dp;
            offset_domain_right[xyz] = (0.5 + params.Nboundary[xyz]) * dp;

            if (params.BC_TYPE == NEW_NO_SLIP) // Nboundary should only be 0 or 1 if we are using the new bc
                offset_recipient[xyz] = 0.1 * params.Nboundary[xyz] * dp;
            else if (params.BC_TYPE == NO_SLIP)
                offset_recipient[xyz] = params.Nboundary[xyz] * dp;
        }
        else // periodic, open ended
        {
            sz[xyz] = params.Nfluid[xyz] + 1;

            offset_domain_left[xyz] = 0.0;
            offset_domain_right[xyz] = dp;
            offset_periodic_fluid[xyz] = 0.75 * dp;
            offset_periodic_recipient[xyz] = 0.85 * dp;
        }
    }

    // Define the boxes
    Box<DIM, real_number> domain({-offset_domain_left[0],
                                  -offset_domain_left[1]},
                                 {params.length[0] + offset_domain_right[0],
                                  params.length[1] + offset_domain_right[1]});

    Box<DIM, real_number> fluid_hole({0.0,
                                      0.0},
                                     {params.length[0] + offset_periodic_fluid[0],
                                      params.length[1] + offset_periodic_fluid[1]});

    real_number wlx = params.waterB;
    real_number wly = params.waterH;

    Box<DIM, real_number> fluid_box({0.0,
                                     0.0},
                                    {wlx,
                                     wly});

    Box<DIM, real_number> recipient({-offset_recipient[0],
                                     -offset_recipient[1]},
                                    {params.length[0] + offset_recipient[0] + offset_periodic_recipient[0],
                                     params.length[1] + offset_recipient[1] + offset_periodic_recipient[1]});

    for (int xyz = 0; xyz < DIM; xyz++) // correct length in periodic case
    {
        if (params.bc[xyz] == PERIODIC)
            params.length[xyz] += dp;
    }

    // extended boundary around the domain, and the processor domain
    Ghost<DIM, real_number> g(params.r_cut);

    // create particle object
    particles vd_loc(0, domain, params.bc, g, DEC_GRAN(128));
    // vd is argument passed as reference we want to fill with particles
    vd = vd_loc;

    // Write constants on file
    // WriteParameters(v_cl, params);

    // place probes
    // if (params.PROBES_ENABLED)
    // {
    //     // we want to place probes  in a vertical line at this locations
    //     Point<DIM, real_number> EndChannel = {params.length[0], 0.0};
    //     Point<DIM, real_number> HalfChannel = {params.length[0] / 2.0, 0.0};
    //     Point<DIM, real_number> HalfHeight = {0.0, params.length[1] / 2.0};
    //     Point<DIM, real_number> VerticalOffset = {0.0, dp};
    //     Point<DIM, real_number> HorizontalOffset = {dp, 0.0};
    //     int k0 = 0;
    //     int kendHeight = params.Nfluid[1] + 1;
    //     int kendWidth = params.Nfluid[0] + 1;

    //     std::vector<Point<DIM, real_number>> ProbePoints; // start points for the PlaceProbes function
    //     std::vector<int> ProbeComponents;                 // component to measure 0 for x 1 for y
    //     std::vector<Point<DIM, real_number>> Offsets;
    //     std::vector<int> maxIters;

    //     if (params.SCENARIO == CAVITY)
    //     {
    //         ProbePoints.push_back(HalfChannel);
    //         ProbePoints.push_back(HalfHeight);

    //         ProbeComponents.push_back(0); // measure x velocity
    //         ProbeComponents.push_back(1); // measure y velocity

    //         Offsets.push_back(VerticalOffset);
    //         Offsets.push_back(HorizontalOffset);

    //         maxIters.push_back(kendHeight);
    //         maxIters.push_back(kendWidth);
    //     }
    //     else
    //     {
    //         ProbePoints.push_back(HalfChannel);
    //         ProbePoints.push_back(EndChannel);

    //         ProbeComponents.push_back(0); // measure x velocity
    //         ProbeComponents.push_back(0); // measure x velocity

    //         Offsets.push_back(VerticalOffset);
    //         Offsets.push_back(VerticalOffset);

    //         maxIters.push_back(kendHeight);
    //         maxIters.push_back(kendHeight);
    //     }

    //     for (unsigned int k = 0; k < ProbePoints.size(); k++)
    //     {
    //         // create probe object
    //         Ghost<DIM, real_number> gp(0);
    //         size_t bc_p[DIM] = {NON_PERIODIC, NON_PERIODIC};
    //         probe_particles vp_loc(0, domain, bc_p, gp, DEC_GRAN(512));
    //         if (ProbeComponents[k] == 0)
    //         {
    //             openfpm::vector<std::string> names_p = {"vx"};
    //             vp_loc.setPropNames(names_p);
    //         }
    //         else if (ProbeComponents[k] == 1)
    //         {
    //             openfpm::vector<std::string> names_p = {"vy"};
    //             vp_loc.setPropNames(names_p);
    //         }

    //         if (v_cl.getProcessUnitID() == 0)
    //         {
    //             PlaceProbes(vp_loc, k0, maxIters[k], ProbePoints[k], Offsets[k]);
    //         }
    //         std::pair<probe_particles, int> tmp = std::make_pair(vp_loc, ProbeComponents[k]);
    //         vp_vec.push_back(tmp);
    //         params.probe_filenames.push_back("probes_" + std::to_string(k) + "_" + params.filename);
    //     }
    // }

    // Add the obstacle/walls as marker particles only on processor 0
    if (params.BC_TYPE == NEW_NO_SLIP && v_cl.getProcessUnitID() == 0)
    {

        real_number dx_wall_x = dp / refine_factor;
        int Nwall_x = ceil(params.length[0] / dx_wall_x);
        dx_wall_x = params.length[0] / Nwall_x;
        Point<DIM, real_number> X_Offset = {dx_wall_x, 0.0};

        real_number dx_wall_y = dp / refine_factor;
        int Nwall_y = ceil(params.length[1] / dx_wall_y);
        dx_wall_y = params.length[1] / Nwall_y;
        Point<DIM, real_number> Y_Offset = {0.0, dx_wall_y};

        Point<DIM, real_number> LL_corner = {0.0, 0.0};
        Point<DIM, real_number> LR_corner = {params.length[0], 0.0};

        Point<DIM, real_number> UL_corner = {0.0, params.length[1]};

        // Top And Bottom Walls
        AddFlatWallModNewBC(vd, 0, 1, LL_corner, X_Offset, dx_wall_x, {0.0, 0.0}, params.vw_bottom, params, BOUNDARY, {10.0 * 1.0, 10.0 * 1.0}, 0.0);
        AddFlatWallModNewBC(vd, Nwall_x, Nwall_x + 1, LL_corner, X_Offset, dx_wall_x, {0.0, 0.0}, params.vw_bottom, params, BOUNDARY, {10.0 * -1.0, 10.0 * 1.0}, 0.0);
        AddFlatWallModNewBC(vd, 1, Nwall_x, LL_corner, X_Offset, dx_wall_x, {0.0, 0.0}, params.vw_bottom, params, BOUNDARY, {0.0, 10.0 * 1.0}, 0.0);

        AddFlatWallModNewBC(vd, 0, 1, UL_corner, X_Offset, dx_wall_x, {0.0, 0.0}, params.vw_top, params, BOUNDARY, {10.0 * 1.0, 10.0 * -1.0}, 0.0);
        AddFlatWallModNewBC(vd, Nwall_x, Nwall_x + 1, UL_corner, X_Offset, dx_wall_x, {0.0, 0.0}, params.vw_top, params, BOUNDARY, {-10.0 * 1.0, 10.0 * -1.0}, 0.0);
        AddFlatWallModNewBC(vd, 1, Nwall_x, UL_corner, X_Offset, dx_wall_x, {0.0, 0.0}, params.vw_top, params, BOUNDARY, {0.0, 10.0 * -1.0}, 0.0);

        // Left And Right Walls
        AddFlatWallModNewBC(vd, 1, Nwall_y, LL_corner, Y_Offset, dx_wall_y, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, {10.0 * 1.0, 0.0}, 0.0);
        AddFlatWallModNewBC(vd, 1, Nwall_y, LR_corner, Y_Offset, dx_wall_y, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, {10.0 * -1.0, 0.0}, 0.0);
    }

    // return an iterator to the fluid particles to add to vd
    auto fluid_it = DrawParticles::DrawBox(vd, sz, domain, fluid_box);

    // for each particle inside the fluid box ...
    while (fluid_it.isNext())
    {

        Point<DIM, real_number> iterator_position = fluid_it.get();

        // ... add a particle ...
        vd.add();
        vd.template getLastProp<vd0_type>() = FLUID;
        vd.template getLastProp<vd10_omega>() = 0.0;
        for (int xyz = 0; xyz < DIM; xyz++)
        {
            vd.template getLastProp<vd4_velocity>()[xyz] = 0.0;
            vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
        }

        // Set properties
        vd.template getLastProp<vd1_rho>() = params.rho0;
        vd.template getLastProp<vd2_pressure>() = 0.0;
        vd.template getLastProp<vd3_drho>() = 0.0;

        for (int xyz = 0; xyz < DIM; xyz++)
        {
            vd.getLastPos()[xyz] = iterator_position.get(xyz);
            vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
            vd.template getLastProp<vd8_normal>()[xyz] = 0.0;
        }

        vd.template getLastProp<vd9_volume>()[0] = dp;
        vd.template getLastProp<vd9_volume>()[1] = 0.0;
        vd.template getLastProp<vd9_volume>()[2] = 0.0;

        // next fluid particle
        ++fluid_it;
    }

    // Now place solid walls using iterators (only for OLD BC)

    if (params.BC_TYPE == NO_SLIP)
    {

        openfpm::vector<Box<DIM, real_number>> holes;
        holes.add(fluid_hole);
        Box<DIM, real_number> hole_box = holes.get(0);
        auto bound_box = DrawParticles::DrawSkin(vd, sz, domain, holes, recipient);

        if (params.bc[0] != PERIODIC || params.bc[1] != PERIODIC) // no walls in all periodic scenario
        {
            while (bound_box.isNext())
            {
                Point<DIM, real_number> position = bound_box.get();

                // periodic bc, with no boundary particles in y direction has a bug, it puts 3 extra particles outside in the y direction
                // When running on multiple cores, with this we check if particle is outside the recipient box
                // Another bug places boundary particles in the correct plane, but inside the fluid box;
                // if (bc[0] == PERIODIC && position.get(0) > dp / 2.0 && position.get(0) < length[0] - dp / 2.0)
                // {
                // 	++bound_box;
                // 	continue;
                // }

                if (!recipient.isInside((position)))
                {
                    ++bound_box;
                    continue;
                }
                if (hole_box.isInside(position))
                {
                    ++bound_box;
                    continue;
                }

                // if (params.BC_TYPE == NEW_NO_SLIP && (params.bc[0] == NON_PERIODIC && params.bc[1] == NON_PERIODIC))
                // {
                // 	// Check if x and z coordinates are multiples of dp, keep multiples, discard the rest
                // 	real_number remx = fmod(position.get(0), dp);
                // 	real_number remz = fmod(position.get(1), dp);
                // 	real_number tol = 0.5 * dp * 10e-2;

                // 	if (remx > tol && remx < dp - tol)
                // 	{
                // 		++bound_box;
                // 		continue;
                // 	}
                // 	if (remz > tol && remz < dp - tol)
                // 	{
                // 		++bound_box;
                // 		continue;
                // 	}
                // }
                vd.add();

                vd.template getLastProp<vd0_type>() = BOUNDARY;
                vd.template getLastProp<vd1_rho>() = params.rho0;
                vd.template getLastProp<vd2_pressure>() = 0.0;
                vd.template getLastProp<vd3_drho>() = 0.0;

                for (int xyz = 0; xyz < DIM; xyz++)
                {
                    vd.getLastPos()[xyz] = bound_box.get().get(xyz);
                    vd.template getLastProp<vd6_force>()[xyz] = 0.0;
                    vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
                    vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
                    vd.template getLastProp<vd8_normal>()[xyz] = 0.0;
                    if (position.get(1) < dp / 4.0) // bottom wall
                    {
                        vd.template getLastProp<vd4_velocity>()[xyz] = params.vw_bottom[xyz];
                    }
                    else if (position.get(1) > params.length[1] - dp / 4.0) // top wall
                    {
                        vd.template getLastProp<vd4_velocity>()[xyz] = params.vw_top[xyz];
                    }
                }

                vd.template getLastProp<vd9_volume>()[0] = dp;

                ++bound_box;
            }
        }
    }
}

void CreateParticleGeometryDamBreakAdj(particles &vd, std::vector<std::pair<probe_particles, int>> &vp_vec, Parameters &params, AuxiliarParameters &auxParams)
{

    Vcluster<> &v_cl = create_vcluster();

    // Non periodic situation grid of 5 fluid particles and 3 boundary particles
    // We need a virtual grid of 5 + 2*(3+1) particles,
    // therefore the domain is discretized with 13 grid points,
    // when we use DrawParticles::DrawBox we will draw only the particles at the grid positons strictly inside the box,
    // the () repesent the recipient box, and the || represent the fluid box, we can see how this distribution places exactly 5 fluid particles inside and 3 boundary particles
    //           D-(-o--o--o-|-x--x--x--x--x--|-o-o-o-)-D
    // D: domain, o: boundary, x: fluid, --: dp distance
    // in a periodic situation we have the following
    // .....--x--x--D-|-x--x--x--x--x--|-D--x--x--......
    // therefore we need a grid of 5 + 2 particles, and the domain is discretized with 7 grid points

    // Size of the virtual cartesian grid that defines where to place the particles
    size_t sz[DIM];

    real_number dp = params.dp;
    real_number refine_factor = params.rf;

    // Now define the iterator boxes
    // We define the boxes in terms of offstes with respect to the fluid box that goes from 0 to length
    real_number offset_domain_left[DIM] = {0.0};
    real_number offset_domain_right[DIM] = {0.0};
    real_number offset_recipient[DIM] = {0.0};
    real_number offset_periodic_fluid[DIM] = {0.0};
    real_number offset_periodic_recipient[DIM] = {0.0};

    for (int xyz = 0; xyz < DIM; xyz++)
    {
        if (params.bc[xyz] == NON_PERIODIC) // non periodic, fluid covered by boundary
        {
            sz[xyz] = params.Nfluid[xyz] + 2 * (params.Nboundary[xyz] + 1);
            offset_domain_left[xyz] = (0.5 + params.Nboundary[xyz]) * dp;
            offset_domain_right[xyz] = (0.5 + params.Nboundary[xyz]) * dp;

            if (params.BC_TYPE == NEW_NO_SLIP) // Nboundary should only be 0 or 1 if we are using the new bc
                offset_recipient[xyz] = 0.1 * params.Nboundary[xyz] * dp;
            else if (params.BC_TYPE == NO_SLIP)
                offset_recipient[xyz] = params.Nboundary[xyz] * dp;
        }
        else // periodic, open ended
        {
            sz[xyz] = params.Nfluid[xyz] + 1;

            offset_domain_left[xyz] = 0.0;
            offset_domain_right[xyz] = dp;
            offset_periodic_fluid[xyz] = 0.75 * dp;
            offset_periodic_recipient[xyz] = 0.85 * dp;
        }
    }

    // Define the boxes
    real_number auxiliary_tank_width = 22.0 * dp;
    real_number auxiliary_tank_height = 6.0 * dp;

    sz[0] += 44;

    Box<DIM, real_number> domain({-offset_domain_left[0],
                                  -offset_domain_left[1]},
                                 {params.length[0] + 2.0 * auxiliary_tank_width + offset_domain_right[0],
                                  params.length[1] + offset_domain_right[1]});

    Box<DIM, real_number> fluid_hole({0.0,
                                      0.0},
                                     {params.length[0] + offset_periodic_fluid[0],
                                      params.length[1] + offset_periodic_fluid[1]});

    real_number wlx = params.waterB;
    real_number wly = params.waterH;

    Box<DIM, real_number> fluid_box({0.0,
                                     0.0},
                                    {wlx,
                                     wly});

    Box<DIM, real_number> second_fluid_box = {{params.length[0] + dp + auxiliary_tank_width,
                                               auxiliary_tank_height},
                                              {params.length[0] + 2.0 * auxiliary_tank_width + offset_periodic_fluid[0],
                                               params.length[1] + offset_periodic_fluid[1] - 2.0 * auxiliary_tank_height}};

    Box<DIM, real_number> recipient({-offset_recipient[0],
                                     -offset_recipient[1]},
                                    {params.length[0] + offset_recipient[0] + offset_periodic_recipient[0],
                                     params.length[1] + offset_recipient[1] + offset_periodic_recipient[1]});

    for (int xyz = 0; xyz < DIM; xyz++) // correct length in periodic case
    {
        if (params.bc[xyz] == PERIODIC)
            params.length[xyz] += dp;
    }

    // extended boundary around the domain, and the processor domain
    Ghost<DIM, real_number> g(params.r_cut);

    // create particle object
    particles vd_loc(0, domain, params.bc, g, DEC_GRAN(128));
    // vd is argument passed as reference we want to fill with particles
    vd = vd_loc;

    // Add the obstacle/walls as marker particles only on processor 0
    if (params.BC_TYPE == NEW_NO_SLIP && v_cl.getProcessUnitID() == 0)
    {

        real_number dx_wall_x = dp / refine_factor;
        int Nwall_x = ceil(params.length[0] / dx_wall_x);
        dx_wall_x = params.length[0] / Nwall_x;
        Point<DIM, real_number> X_Offset = {dx_wall_x, 0.0};

        real_number dx_wall_y = dp / refine_factor;
        int Nwall_y = ceil(params.length[1] / dx_wall_y);
        dx_wall_y = params.length[1] / Nwall_y;
        Point<DIM, real_number> Y_Offset = {0.0, dx_wall_y};

        Point<DIM, real_number> LL_corner = {0.0, 0.0};
        Point<DIM, real_number> LR_corner = {params.length[0], 0.0};
        Point<DIM, real_number> UL_corner = {0.0, params.length[1]};

        Point<DIM, real_number> ColumnLL = {params.length[0] + dp + auxiliary_tank_width, auxiliary_tank_height};
        Point<DIM, real_number> ColumnLR = {params.length[0] + 2.0 * auxiliary_tank_width, auxiliary_tank_height};
        Point<DIM, real_number> ColumnUL = {params.length[0] + dp + auxiliary_tank_width, params.length[1] - auxiliary_tank_height};

        int Nwall_x_small = ceil(auxiliary_tank_width / dx_wall_x);
        int Nwall_y_small = ceil((params.length[1] - 2 * auxiliary_tank_height) / dx_wall_y);
        real_number fac = 0.5;
        // Top And Bottom Walls
        AddFlatWallModNewBC(vd, 0, 1, LL_corner, X_Offset, dx_wall_x, {0.0, 0.0}, params.vw_bottom, params, BOUNDARY, {fac * 1.0, fac * 1.0}, 0.0, 0.0);
        AddFlatWallModNewBC(vd, Nwall_x, Nwall_x + 1, LL_corner, X_Offset, dx_wall_x, {0.0, 0.0}, params.vw_bottom, params, BOUNDARY, {fac * -1.0, fac * 1.0}, 0.0, 0.0);
        AddFlatWallModNewBC(vd, 1, Nwall_x, LL_corner, X_Offset, dx_wall_x, {0.0, 0.0}, params.vw_bottom, params, BOUNDARY, {0.0, fac * 1.0}, 0.0, 0.0);

        AddFlatWallModNewBC(vd, 0, 1, UL_corner, X_Offset, dx_wall_x, {0.0, 0.0}, params.vw_top, params, BOUNDARY, {fac * 1.0, fac * -1.0}, 0.0), 0.0;
        AddFlatWallModNewBC(vd, Nwall_x, Nwall_x + 1, UL_corner, X_Offset, dx_wall_x, {0.0, 0.0}, params.vw_top, params, BOUNDARY, {-fac * 1.0, fac * -1.0}, 0.0, 0.0);
        AddFlatWallModNewBC(vd, 1, Nwall_x, UL_corner, X_Offset, dx_wall_x, {0.0, 0.0}, params.vw_top, params, BOUNDARY, {0.0, fac * -1.0}, 0.0, 0.0);

        // Left And Right Walls
        AddFlatWallModNewBC(vd, 1, Nwall_y, LL_corner, Y_Offset, dx_wall_y, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, {fac * 1.0, 0.0}, 0.0, 0.0);
        AddFlatWallModNewBC(vd, 1, Nwall_y, LR_corner, Y_Offset, dx_wall_y, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, {fac * -1.0, 0.0}, 0.0, 0.0);

        // Auxiliary Tank
        AddFlatWallModNewBC(vd, 0, 1, ColumnLL, X_Offset, dx_wall_x, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, {fac * 1.0, fac * 1.0}, 0.0, 0.0);
        AddFlatWallModNewBC(vd, 1, Nwall_x_small - 1, ColumnLL, X_Offset, dx_wall_x, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, {0.0, fac * 1.0}, 0.0, 0.0);
        AddFlatWallModNewBC(vd, Nwall_x_small - 1, Nwall_x_small, ColumnLL, X_Offset, dx_wall_x, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, {fac * -1.0, fac * 1.0}, 0.0, 0.0);

        AddFlatWallModNewBC(vd, 0, 1, ColumnUL, X_Offset, dx_wall_x, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, {fac * 1.0, -fac * 1.0}, 0.0, 0.0);
        AddFlatWallModNewBC(vd, 1, Nwall_x_small - 1, ColumnUL, X_Offset, dx_wall_x, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, {0.0, -fac * 1.0}, 0.0, 0.0);
        AddFlatWallModNewBC(vd, Nwall_x_small - 1, Nwall_x_small, ColumnUL, X_Offset, dx_wall_x, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, {fac * -1.0, -fac * 1.0}, 0.0, 0.0);

        AddFlatWallModNewBC(vd, 1, Nwall_y_small, ColumnLL, Y_Offset, dx_wall_y, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, {fac * 1.0, 0.0}, 0.0, 0.0);
        AddFlatWallModNewBC(vd, 1, Nwall_y_small, ColumnLR, Y_Offset, dx_wall_y, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, {fac * -1.0, 0.0}, 0.0, 0.0);
    }

    // return an iterator to the fluid particles to add to vd
    auto fluid_it = DrawParticles::DrawBox(vd, sz, domain, fluid_box);
    auto fluid_it2 = DrawParticles::DrawBox(vd, sz, domain, second_fluid_box);

    // for each particle inside the fluid box ...
    while (fluid_it.isNext())
    {

        Point<DIM, real_number> iterator_position = fluid_it.get();

        // ... add a particle ...
        vd.add();
        vd.template getLastProp<vd0_type>() = FLUID;
        vd.template getLastProp<vd10_omega>() = 0.0;
        for (int xyz = 0; xyz < DIM; xyz++)
        {
            vd.template getLastProp<vd4_velocity>()[xyz] = 0.0;
            vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
        }

        // Set properties
        vd.template getLastProp<vd1_rho>() = params.rho0;
        vd.template getLastProp<vd2_pressure>() = 0.0;
        vd.template getLastProp<vd3_drho>() = 0.0;

        for (int xyz = 0; xyz < DIM; xyz++)
        {
            vd.getLastPos()[xyz] = iterator_position.get(xyz);
            vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
            vd.template getLastProp<vd8_normal>()[xyz] = 0.0;
        }

        vd.template getLastProp<vd9_volume>()[0] = dp;
        vd.template getLastProp<vd9_volume>()[1] = 0.0;
        vd.template getLastProp<vd9_volume>()[2] = 0.0;

        // next fluid particle
        ++fluid_it;
    }
    // for each particle inside the fluid box ...
    while (fluid_it2.isNext())
    {

        Point<DIM, real_number> iterator_position = fluid_it2.get();

        // ... add a particle ...
        vd.add();
        vd.template getLastProp<vd0_type>() = FLUID;
        vd.template getLastProp<vd10_omega>() = 0.0;
        for (int xyz = 0; xyz < DIM; xyz++)
        {
            vd.template getLastProp<vd4_velocity>()[xyz] = 0.0;
            vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
        }

        // Set properties
        vd.template getLastProp<vd1_rho>() = params.rho0;
        vd.template getLastProp<vd2_pressure>() = 0.0;
        vd.template getLastProp<vd3_drho>() = 0.0;

        for (int xyz = 0; xyz < DIM; xyz++)
        {
            vd.getLastPos()[xyz] = iterator_position.get(xyz);
            vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
            vd.template getLastProp<vd8_normal>()[xyz] = 0.0;
        }

        vd.template getLastProp<vd9_volume>()[0] = dp;
        vd.template getLastProp<vd9_volume>()[1] = 0.0;
        vd.template getLastProp<vd9_volume>()[2] = 0.0;

        // next fluid particle
        ++fluid_it2;
    }

    // Create a Cell list object for CPU calculations
    auto NN = vd.getCellList(params.r_cut);
    vd.updateCellList(NN);

    CalcFluidVec(vd, NN, params);
    vd.ghost_get<vd0_type, vd1_rho, vd2_pressure, vd4_velocity, vd5_velocity_t, vd6_force, vd7_force_t, vd8_normal, vd9_volume, vd10_omega, vd11_vorticity>();

    CalcNormalVec(vd, NN, params);
    vd.ghost_get<vd0_type, vd1_rho, vd2_pressure, vd4_velocity, vd5_velocity_t, vd6_force, vd7_force_t, vd8_normal, vd9_volume, vd10_omega, vd11_vorticity>();

    CalcCurvature(vd, NN, params);
    vd.ghost_get<vd0_type, vd1_rho, vd2_pressure, vd4_velocity, vd5_velocity_t, vd6_force, vd7_force_t, vd8_normal, vd9_volume, vd10_omega, vd11_vorticity>();

    CalcVolume(vd, params.dp);
    vd.ghost_get<vd0_type, vd1_rho, vd2_pressure, vd4_velocity, vd5_velocity_t, vd6_force, vd7_force_t, vd8_normal, vd9_volume, vd10_omega, vd11_vorticity>();

    vd.write_frame("frametest", 0, 0.0, VTK_WRITER);
    // now move aux tank beside the dam break
    auto it = vd.getDomainIterator();
    while (it.isNext())
    {
        auto akey = it.get();

        Point<DIM, real_number> position = vd.getPos(akey);
        real_number xthreshold = params.length[0] + 0.5 * auxiliary_tank_width;
        if (position.get(0) > xthreshold)
        {
            vd.getPos(akey)[0] -= (auxiliary_tank_width + 0.99 * dp);
        }

        ++it;
    }
}

void CreateParticleGeometryCavity(particles &vd, std::vector<std::pair<probe_particles, int>> &vp_vec, Obstacle *&obstacle_ptr, Parameters &params, AuxiliarParameters &auxParams)
{

    Vcluster<> &v_cl = create_vcluster();
    obstacle_ptr = new EmptyObstacle(params);

    // Size of the virtual cartesian grid that defines where to place the particles
    size_t sz[DIM];
    real_number dp = params.dp;

    // Now define the iterator boxes
    // We define the boxes in terms of offstes with respect to the fluid box that goes from 0 to length
    real_number offset_domain_left[DIM] = {0.0};
    real_number offset_domain_right[DIM] = {0.0};
    real_number offset_recipient[DIM] = {0.0};
    // Nboundary[0] = 3;
    // Nboundary[1] = 3;

    sz[0] = params.Nfluid[0] + 2 * (params.Nboundary[0] + 1) + 6;
    sz[1] = params.Nfluid[1] + 2 * (params.Nboundary[1] + 1);

    offset_domain_left[0] = (3.5 + params.Nboundary[0]) * dp;
    offset_domain_left[1] = (0.5 + params.Nboundary[1]) * dp;

    offset_domain_right[0] = (3.5 + params.Nboundary[0]) * dp;
    offset_domain_right[1] = (0.5 + params.Nboundary[1]) * dp;

    if (params.BC_TYPE == NEW_NO_SLIP) // Nboundary should only be 0 or 1 if we are using the new bc
    {
        offset_recipient[0] = 0.1 * params.Nboundary[0] * dp;
        offset_recipient[1] = 0.1 * params.Nboundary[1] * dp;
    }
    else if (params.BC_TYPE == NO_SLIP)
    {
        offset_recipient[0] = params.Nboundary[0] * dp;
        offset_recipient[1] = params.Nboundary[1] * dp;
    }

    // Define the boxes
    Box<DIM, real_number> domain({-offset_domain_left[0],
                                  -offset_domain_left[1]},
                                 {params.length[0] + offset_domain_right[0],
                                  params.length[1] + offset_domain_right[1]});

    Box<DIM, real_number> fluid_box({0.0,
                                     0.0},
                                    {params.length[0],
                                     params.length[1]});

    Box<DIM, real_number> recipient({-offset_recipient[0],
                                     -offset_recipient[1]},
                                    {params.length[0] + offset_recipient[0],
                                     params.length[1] + offset_recipient[1]});

    // for (int xyz = 0; xyz < DIM; xyz++) // correct length in periodic case
    // {
    //     if (params.bc[xyz] == PERIODIC)
    //         params.length[xyz] += dp;
    // }

    // extended boundary around the domain, and the processor domain
    Ghost<DIM, real_number> g(params.r_cut);

    // create particle object
    particles vd_loc(0, domain, params.bc, g, DEC_GRAN(128));
    // vd is argument passed as reference we want to fill with particles
    vd = vd_loc;

    // place probes
    if (params.PROBES_ENABLED)
    {
        // we want to place probes in a vertical line at this locations
        Point<DIM, real_number> HalfChannel = {params.length[0] / 2.0, 0.0};
        Point<DIM, real_number> HalfHeight = {0.0, params.length[1] / 2.0};
        Point<DIM, real_number> VerticalOffset = {0.0, dp};
        Point<DIM, real_number> HorizontalOffset = {dp, 0.0};
        int k0 = 0;
        int kendHeight = params.Nfluid[1] + 1;
        int kendWidth = params.Nfluid[0] + 1;

        std::vector<Point<DIM, real_number>> ProbePoints; // start points for the PlaceProbes function
        std::vector<int> ProbeComponents;                 // velocity component to measure 0 for x 1 for y
        std::vector<Point<DIM, real_number>> Offsets;
        std::vector<int> maxIters;

        std::vector<int> FixedProbeIndices_horizontal = {k0, kendWidth - 1};
        std::vector<int> FixedProbeIndices_vertical = {k0, kendHeight - 1};
        std::vector<real_number> FixedProbeValues_horizontal = {0.0, 0.0};
        std::vector<real_number> FixedProbeValues_vertical = {0.0, params.vw_top[0]};

        std::vector<std::vector<int>> FixedProbeIndices;
        std::vector<std::vector<real_number>> FixedProbeValues;

        if (params.SCENARIO == CAVITY)
        {
            ProbePoints.push_back(HalfChannel);
            ProbePoints.push_back(HalfHeight);

            ProbeComponents.push_back(0); // measure x velocity
            ProbeComponents.push_back(1); // measure y velocity

            Offsets.push_back(VerticalOffset);
            Offsets.push_back(HorizontalOffset);

            maxIters.push_back(kendHeight);
            maxIters.push_back(kendWidth);

            FixedProbeIndices.push_back(FixedProbeIndices_vertical);
            FixedProbeIndices.push_back(FixedProbeIndices_horizontal);
            FixedProbeValues.push_back(FixedProbeValues_vertical);
            FixedProbeValues.push_back(FixedProbeValues_horizontal);
        }

        for (unsigned int k = 0; k < ProbePoints.size(); k++)
        {
            // create probe object
            Ghost<DIM, real_number> gp(0);
            size_t bc_p[DIM] = {NON_PERIODIC, NON_PERIODIC};
            probe_particles vp_loc(0, domain, bc_p, gp, DEC_GRAN(512));

            if (v_cl.getProcessUnitID() == 0)
            {
                PlaceProbes(vp_loc, k0, maxIters[k], ProbePoints[k], Offsets[k], obstacle_ptr, FixedProbeIndices[k], FixedProbeValues[k]);
            }
            if (ProbeComponents[k] == 0) // measuring x velocity
            {
                openfpm::vector<std::string> names_p({"type", "vx"});
                vp_loc.setPropNames(names_p);
            }
            else if (ProbeComponents[k] == 1) // measuring y velocity
            {
                openfpm::vector<std::string> names_p({"type", "vy"});
                vp_loc.setPropNames(names_p);
            }
            std::pair<probe_particles, int> tmp = std::make_pair(vp_loc, ProbeComponents[k]);
            vp_vec.push_back(tmp);
            auxParams.probe_filenames.push_back("probes_" + std::to_string(k) + "_" + auxParams.filename);
        }
    }

    // Add the obstacle/walls as marker particles only on processor 0
    if (params.BC_TYPE == NEW_NO_SLIP && v_cl.getProcessUnitID() == 0)
    {
        real_number refine_factor = params.rf;
        real_number dx_wall_x = dp / refine_factor;
        int Nwall_x_bot = ceil(params.length[0] / dx_wall_x);
        int Nwall_x_top = ceil((params.length[0] + 9.0 * dp) / dx_wall_x);

        dx_wall_x = params.length[0] / Nwall_x_bot;
        Point<DIM, real_number> X_Offset = {dx_wall_x, 0.0};

        real_number dx_wall_y = dp / refine_factor;
        int Nwall_y = ceil(params.length[1] / dx_wall_y);
        dx_wall_y = params.length[1] / Nwall_y;
        Point<DIM, real_number> Y_Offset = {0.0, dx_wall_y};

        Point<DIM, real_number> LL_corner = {0.0, 0.0};
        Point<DIM, real_number> LR_corner = {params.length[0], 0.0};

        Point<DIM, real_number> UL_corner = {-4.0 * dp, params.length[1]};

        // Top And Bottom Walls
        AddFlatWallNewBC(vd, 0, Nwall_x_bot + 1, LL_corner, X_Offset, dx_wall_x, {0.0, 0.0}, params.vw_bottom, params, BOUNDARY, 0.0);
        AddFlatWallModNewBC(vd, 0, Nwall_x_top + 1, UL_corner, X_Offset, dx_wall_x, {0.0, 0.0}, params.vw_top, params, BOUNDARY, {0.0, -10.0}, 1.0, 0.0);

        // Left And Right Walls
        real_number r_cut_int = ceil(params.r_cut / dx_wall_y); // r_cut expressed in number of particles ( ceil to get an integer )
        // from 1 to N_wall_y - r_cut_int add normal wall
        AddFlatWallNewBC(vd, 1, Nwall_y - r_cut_int, LL_corner, Y_Offset, dx_wall_y, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, 0.0);
        AddFlatWallNewBC(vd, 1, Nwall_y - r_cut_int, LR_corner, Y_Offset, dx_wall_y, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, 0.0);
        // From N_wall_y - r_cut_int to N_wall_y add wall with prescribed normal and curvature
        AddFlatWallModNewBC(vd, Nwall_y - r_cut_int, Nwall_y + 1, LL_corner, Y_Offset, dx_wall_y, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, {10.0, 0.0}, 1.0, 0.0);
        AddFlatWallModNewBC(vd, Nwall_y - r_cut_int, Nwall_y + 1, LR_corner, Y_Offset, dx_wall_y, {0.0, 0.0}, {0.0, 0.0}, params, BOUNDARY, {-10.0, 0.0}, 1.0, 0.0);
    }

    // return an iterator to the fluid particles to add to vd
    auto fluid_it = DrawParticles::DrawBox(vd, sz, domain, fluid_box);

    // for each particle inside the fluid box ...
    while (fluid_it.isNext())
    {

        Point<DIM, real_number> iterator_position = fluid_it.get();

        // ... add a particle ...
        vd.add();
        vd.template getLastProp<vd0_type>() = FLUID;
        vd.template getLastProp<vd10_omega>() = 0.0;
        for (int xyz = 0; xyz < DIM; xyz++)
        {
            vd.template getLastProp<vd4_velocity>()[xyz] = 0.0;
            vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
        }

        // Set properties
        vd.template getLastProp<vd1_rho>() = params.rho0;
        vd.template getLastProp<vd2_pressure>() = 0.0;
        vd.template getLastProp<vd3_drho>() = 0.0;

        for (int xyz = 0; xyz < DIM; xyz++)
        {
            vd.getLastPos()[xyz] = iterator_position.get(xyz);
            vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
            vd.template getLastProp<vd8_normal>()[xyz] = 0.0;
        }

        vd.template getLastProp<vd9_volume>()[0] = dp;
        vd.template getLastProp<vd9_volume>()[1] = 0.0;
        vd.template getLastProp<vd9_volume>()[2] = 0.0;

        // next fluid particle
        ++fluid_it;
    }

    // Now place solid walls using iterators (only for OLD BC)

    if (params.BC_TYPE == NO_SLIP)
    {

        openfpm::vector<Box<DIM, real_number>> holes;
        holes.add(fluid_box);
        Box<DIM, real_number> hole_box = holes.get(0);
        auto bound_box = DrawParticles::DrawSkin(vd, sz, domain, holes, recipient);

        while (bound_box.isNext())
        {
            Point<DIM, real_number> position = bound_box.get();

            // periodic bc, with no boundary particles in y direction has a bug, it puts 3 extra particles outside in the y direction
            // When running on multiple cores, with this we check if particle is outside the recipient box
            // Another bug places boundary particles in the correct plane, but inside the fluid box;
            // if (bc[0] == PERIODIC && position.get(0) > dp / 2.0 && position.get(0) < length[0] - dp / 2.0)
            // {
            // 	++bound_box;
            // 	continue;
            // }

            if (!recipient.isInside((position)))
            {
                ++bound_box;
                continue;
            }
            if (hole_box.isInside(position))
            {
                ++bound_box;
                continue;
            }

            vd.add();

            vd.template getLastProp<vd0_type>() = BOUNDARY;
            vd.template getLastProp<vd1_rho>() = params.rho0;
            vd.template getLastProp<vd2_pressure>() = 0.0;
            vd.template getLastProp<vd3_drho>() = 0.0;

            for (int xyz = 0; xyz < DIM; xyz++)
            {
                vd.getLastPos()[xyz] = bound_box.get().get(xyz);
                vd.template getLastProp<vd6_force>()[xyz] = 0.0;
                vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
                vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
                vd.template getLastProp<vd8_normal>()[xyz] = 0.0;
                if (position.get(1) < dp / 4.0) // bottom wall
                {
                    vd.template getLastProp<vd4_velocity>()[xyz] = params.vw_bottom[xyz];
                }
                else if (position.get(1) > params.length[1] - dp / 4.0) // top wall
                {
                    vd.template getLastProp<vd4_velocity>()[xyz] = params.vw_top[xyz];
                }
            }

            vd.template getLastProp<vd9_volume>()[0] = dp;
            vd.template getLastProp<vd9_volume>()[1] = 0.0;
            vd.template getLastProp<vd9_volume>()[2] = 0.0;
            vd.template getLastProp<vd11_vorticity>() = 0.0;
            vd.template getLastProp<vd12_vel_red>() = 0.0;
            vd.template getLastProp<vd13_force_red_x>() = 0.0;
            vd.template getLastProp<vd14_force_red_y>() = 0.0;

            ++bound_box;
        }

        if (v_cl.getProcessUnitID() == 0)
        {
            // RIGHT CORNER
            const Point<DIM, real_number> Corner = {params.length[0] + 2.5 * dp,
                                                    params.length[1] + 0.5 * dp};

            // Manually place chunck of particles to fill periodicity of top wall
            for (int i = 0; i < 3; i++)
            {
                for (int j = 0; j < 3; j++)
                {

                    Point<DIM, real_number> offset = {(j + 1) * dp, (i)*dp};
                    Point<DIM, real_number> position = Corner + offset;

                    vd.add();

                    vd.template getLastProp<vd0_type>() = BOUNDARY;
                    vd.template getLastProp<vd1_rho>() = params.rho0;
                    vd.template getLastProp<vd2_pressure>() = 0.0;
                    vd.template getLastProp<vd3_drho>() = 0.0;

                    for (int xyz = 0; xyz < DIM; xyz++)
                    {
                        vd.getLastPos()[xyz] = position.get(xyz);
                        vd.template getLastProp<vd6_force>()[xyz] = 0.0;
                        vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
                        vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
                        vd.template getLastProp<vd8_normal>()[xyz] = 0.0;

                        vd.template getLastProp<vd4_velocity>()[xyz] = params.vw_top[xyz];
                    }

                    vd.template getLastProp<vd9_volume>()[0] = dp;
                }
            }

            // LEFT CORNER
            const Point<DIM, real_number> CornerUL = {0.0 - 2.5 * dp,
                                                      params.length[1] + 0.5 * dp};

            // Manually place chunck of particles to fill periodicity of top wall
            for (int i = 0; i < 3; i++)
            {
                for (int j = 0; j < 4; j++)
                {

                    Point<DIM, real_number> offset = {-(j + 1) * dp, (i)*dp};
                    Point<DIM, real_number> position = CornerUL + offset;

                    vd.add();

                    vd.template getLastProp<vd0_type>() = BOUNDARY;
                    vd.template getLastProp<vd1_rho>() = params.rho0;
                    vd.template getLastProp<vd2_pressure>() = 0.0;
                    vd.template getLastProp<vd3_drho>() = 0.0;

                    for (int xyz = 0; xyz < DIM; xyz++)
                    {
                        vd.getLastPos()[xyz] = position.get(xyz);
                        vd.template getLastProp<vd6_force>()[xyz] = 0.0;
                        vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
                        vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
                        vd.template getLastProp<vd8_normal>()[xyz] = 0.0;

                        vd.template getLastProp<vd4_velocity>()[xyz] = params.vw_top[xyz];
                    }

                    vd.template getLastProp<vd9_volume>()[0] = dp;
                }
            }
        }
    }
}

void CreateParticleGeometrySphere(particles &vd, std::vector<std::pair<probe_particles, int>> &vp_vec, Obstacle *&obstacle_ptr, Parameters &params, AuxiliarParameters &auxParams)
{

    Vcluster<> &v_cl = create_vcluster();

    // Non periodic situation grid of 5 fluid particles and 3 boundary particles
    // We need a virtual grid of 5 + 2*(3+1) particles,
    // therefore the domain is discretized with 13 grid points,
    // when we use DrawParticles::DrawBox we will draw only the particles at the grid positons strictly inside the box,
    // the () repesent the recipient box, and the || represent the fluid box, we can see how this distribution places exactly 5 fluid particles inside and 3 boundary particles
    //           D-(-o--o--o-|-x--x--x--x--x--|-o-o-o-)-D
    // D: domain, o: boundary, x: fluid, --: dp distance
    // in a periodic situation we have the following
    // .....--x--x--D-|-x--x--x--x--x--|-D--x--x--......
    // therefore we need a grid of 5 + 2 particles, and the domain is discretized with 7 grid points

    // Size of the virtual cartesian grid that defines where to place the particles
    size_t sz[DIM];

    obstacle_ptr = new EmptyObstacle(params);
    real_number dp = params.dp;
    // Initialize obstacle in scenarios where needed
    if (params.SCENARIO == SPHERE)
        obstacle_ptr = new SphereObstacle(params.ObstacleBase, params.ObstacleCenter, params, params.ObstacleVelocity, params.ObstacleOmega, params.rf, true);

    real_number refine_factor = params.rf;

    // Now define the iterator boxes
    // We define the boxes in terms of offstes with respect to the fluid box that goes from 0 to length
    real_number offset_domain_left[DIM] = {0.0};
    real_number offset_domain_right[DIM] = {0.0};
    real_number offset_recipient[DIM] = {0.0};
    real_number offset_periodic_fluid[DIM] = {0.0};
    real_number offset_periodic_recipient[DIM] = {0.0};

    for (int xyz = 0; xyz < DIM; xyz++)
    {
        if (params.bc[xyz] == NON_PERIODIC) // non periodic, fluid covered by boundary
        {
            sz[xyz] = params.Nfluid[xyz] + 2 * (params.Nboundary[xyz] + 1);
            offset_domain_left[xyz] = (0.5 + params.Nboundary[xyz]) * dp;
            offset_domain_right[xyz] = (0.5 + params.Nboundary[xyz]) * dp;

            if (params.BC_TYPE == NEW_NO_SLIP) // Nboundary should only be 0 or 1 if we are using the new bc
                offset_recipient[xyz] = 0.1 * params.Nboundary[xyz] * dp;
            else if (params.BC_TYPE == NO_SLIP)
                offset_recipient[xyz] = params.Nboundary[xyz] * dp;
        }
        else // periodic, open ended
        {
            sz[xyz] = params.Nfluid[xyz] + 1;

            offset_domain_left[xyz] = 0.0;
            offset_domain_right[xyz] = dp;
            offset_periodic_fluid[xyz] = 0.75 * dp;
            offset_periodic_recipient[xyz] = 0.85 * dp;
        }
    }

    // Define the boxes
    Box<DIM, real_number> domain({-offset_domain_left[0],
                                  -offset_domain_left[1],
                                  -offset_domain_left[2]},
                                 {params.length[0] + offset_domain_right[0],
                                  params.length[1] + offset_domain_right[1],
                                  params.length[2] + offset_domain_right[2]});

    Box<DIM, real_number> fluid_box({0.0,
                                     0.0,
                                     0.0},
                                    {params.length[0] + offset_periodic_fluid[0],
                                     params.length[1] + offset_periodic_fluid[1],
                                     params.length[2] + offset_periodic_fluid[2]});

    Box<DIM, real_number> recipient({-offset_recipient[0],
                                     -offset_recipient[1],
                                     -offset_recipient[2]},
                                    {params.length[0] + offset_recipient[0] + offset_periodic_recipient[0],
                                     params.length[1] + offset_recipient[1] + offset_periodic_recipient[1],
                                     params.length[2] + offset_recipient[2] + offset_periodic_recipient[2]});

    // Will only be used in the new bc
    Box<DIM, real_number> recipient_hole({offset_recipient[0],
                                          offset_recipient[1],
                                          offset_recipient[2]},
                                         {params.length[0] - offset_recipient[0] + offset_periodic_fluid[0],
                                          params.length[1] - offset_recipient[1] + offset_periodic_fluid[1],
                                          params.length[2] - offset_recipient[2] + offset_periodic_fluid[2]});

    for (int xyz = 0; xyz < DIM; xyz++) // correct length in periodic case
    {
        if (params.bc[xyz] == PERIODIC)
            params.length[xyz] += dp;
    }

    // extended boundary around the domain, and the processor domain
    Ghost<DIM, real_number> g(params.r_cut);

    // create particle object
    particles vd_loc(0, domain, params.bc, g, DEC_GRAN(128));
    // vd is argument passed as reference we want to fill with particles
    vd = vd_loc;

    // Add the obstacle/walls as marker particles only on processor 0
    if (params.BC_TYPE == NEW_NO_SLIP && v_cl.getProcessUnitID() == 0)
    {
        // Add obstacle
        obstacle_ptr->AddObstacle(vd);

        // Add walls
        if (params.bc[0] == PERIODIC && params.bc[1] == NON_PERIODIC && params.bc[2] == NON_PERIODIC) // Channel like scenario
        {
            real_number dx_wall = dp / refine_factor;
            int Nwall_x = ceil(params.length[0] / dx_wall);
            int Nwall_y = ceil(params.length[1] / dx_wall);
            int Nwall_z = ceil(params.length[2] / dx_wall);

            dx_wall = params.length[0] / Nwall_x;
            Point<DIM, real_number> X_Offset = {dx_wall, 0.0, 0.0};
            Point<DIM, real_number> Y_Offset = {0.0, dx_wall, 0.0};
            Point<DIM, real_number> Z_Offset = {0.0, 0.0, dx_wall};

            Point<DIM, real_number> LL_corner = {0.0, 0.0, 0.0};
            Point<DIM, real_number> UR_corner = {0.0, params.length[1], params.length[2]};

            AddFlatWallModNewBC3D(vd, 0, Nwall_x, 0, Nwall_z + 1, LL_corner, X_Offset, Z_Offset, dx_wall, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, params, FREE_SLIP_BOUNDARY, {0.0, 0.0, 0.0}, 0.0, 0.0);
            AddFlatWallModNewBC3D(vd, 0, Nwall_x, 1, Nwall_y + 1, LL_corner, X_Offset, Y_Offset, dx_wall, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, params, FREE_SLIP_BOUNDARY, {0.0, 0.0, 0.0}, 0.0, 0.0);

            AddFlatWallModNewBC3D(vd, 0, Nwall_x, 0, Nwall_z, UR_corner, X_Offset, -Z_Offset, dx_wall, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, params, FREE_SLIP_BOUNDARY, {0.0, 0.0, 0.0}, 0.0, 0.0);
            AddFlatWallModNewBC3D(vd, 0, Nwall_x, 1, Nwall_y, UR_corner, X_Offset, -Y_Offset, dx_wall, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, params, FREE_SLIP_BOUNDARY, {0.0, 0.0, 0.0}, 0.0, 0.0);
        }
    }

    // return an iterator to the fluid particles to add to vd
    auto fluid_it = DrawParticles::DrawBox(vd, sz, domain, fluid_box);

    // for each particle inside the fluid box ...
    while (fluid_it.isNext())
    {

        Point<DIM, real_number> iterator_position = fluid_it.get();

        if ((*obstacle_ptr).isInside(iterator_position)) // if inside the obstacle region
        {
            if (params.BC_TYPE == NO_SLIP) // add particle but set it as boundary
            {
                // ... add a particle ...
                vd.add();
                vd.template getLastProp<vd0_type>() = OBSTACLE;
                vd.template getLastProp<vd10_omega>() = (*obstacle_ptr).AngularVelocity_;
                for (int xyz = 0; xyz < DIM; xyz++)
                {
                    vd.template getLastProp<vd4_velocity>()[xyz] = ((*obstacle_ptr).LinearVelocity_).get(xyz);
                    vd.template getLastProp<vd7_force_t>()[xyz] = ((*obstacle_ptr).Centre_).get(xyz);
                }
            }
            else if (params.BC_TYPE == NEW_NO_SLIP) // not add particle because already added
            {
                ++fluid_it;
                continue;
            }
        }
        else // if not inside obstacle at all just add fluid particles
        {
            // ... add a particle ...
            vd.add();
            vd.template getLastProp<vd0_type>() = FLUID;
            vd.template getLastProp<vd10_omega>() = 0.0;
            for (int xyz = 0; xyz < DIM; xyz++)
            {
                vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
            }
        }

        // Set properties
        vd.template getLastProp<vd1_rho>() = params.rho0;
        vd.template getLastProp<vd2_pressure>() = 0.0;
        vd.template getLastProp<vd3_drho>() = 0.0;

        for (int xyz = 0; xyz < DIM; xyz++)
        {
            vd.getLastPos()[xyz] = iterator_position.get(xyz);
            vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
            vd.template getLastProp<vd8_normal>()[xyz] = 0.0;
        }

        vd.template getLastProp<vd4_velocity>()[0] = 0.0;
        vd.template getLastProp<vd4_velocity>()[1] = 0.0;
        vd.template getLastProp<vd4_velocity>()[2] = 0.0;

        vd.template getLastProp<vd9_volume>()[0] = dp;
        vd.template getLastProp<vd9_volume>()[1] = 0.0;
        vd.template getLastProp<vd9_volume>()[2] = 0.0;

        // next fluid particle
        ++fluid_it;
    }

    // Now place solid walls using iterators (only for OLD BC)

    if (params.BC_TYPE == NO_SLIP)
    {

        openfpm::vector<Box<DIM, real_number>> holes;
        holes.add(fluid_box);
        Box<DIM, real_number> hole_box = holes.get(0);
        auto bound_box = DrawParticles::DrawSkin(vd, sz, domain, holes, recipient);

        if (params.bc[0] != PERIODIC || params.bc[1] != PERIODIC) // no walls in all periodic scenario
        {
            while (bound_box.isNext())
            {
                Point<DIM, real_number> position = bound_box.get();

                // periodic bc, with no boundary particles in y direction has a bug, it puts 3 extra particles outside in the y direction
                // When running on multiple cores, with this we check if particle is outside the recipient box
                // Another bug places boundary particles in the correct plane, but inside the fluid box;
                // if (bc[0] == PERIODIC && position.get(0) > dp / 2.0 && position.get(0) < length[0] - dp / 2.0)
                // {
                // 	++bound_box;
                // 	continue;
                // }

                if (!recipient.isInside((position)))
                {
                    ++bound_box;
                    continue;
                }
                if (hole_box.isInside(position))
                {
                    ++bound_box;
                    continue;
                }

                // if (params.BC_TYPE == NEW_NO_SLIP && (params.bc[0] == NON_PERIODIC && params.bc[1] == NON_PERIODIC))
                // {
                // 	// Check if x and z coordinates are multiples of dp, keep multiples, discard the rest
                // 	real_number remx = fmod(position.get(0), dp);
                // 	real_number remz = fmod(position.get(1), dp);
                // 	real_number tol = 0.5 * dp * 10e-2;

                // 	if (remx > tol && remx < dp - tol)
                // 	{
                // 		++bound_box;
                // 		continue;
                // 	}
                // 	if (remz > tol && remz < dp - tol)
                // 	{
                // 		++bound_box;
                // 		continue;
                // 	}
                // }
                vd.add();

                vd.template getLastProp<vd0_type>() = BOUNDARY;
                vd.template getLastProp<vd1_rho>() = params.rho0;
                vd.template getLastProp<vd2_pressure>() = 0.0;
                vd.template getLastProp<vd3_drho>() = 0.0;

                for (int xyz = 0; xyz < DIM; xyz++)
                {
                    vd.getLastPos()[xyz] = bound_box.get().get(xyz);
                    vd.template getLastProp<vd6_force>()[xyz] = 0.0;
                    vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
                    vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
                    vd.template getLastProp<vd8_normal>()[xyz] = 0.0;
                    if (position.get(1) < dp / 4.0) // bottom wall
                    {
                        vd.template getLastProp<vd4_velocity>()[xyz] = params.vw_bottom[xyz];
                    }
                    else if (position.get(1) > params.length[1] - dp / 4.0) // top wall
                    {
                        vd.template getLastProp<vd4_velocity>()[xyz] = params.vw_top[xyz];
                    }
                }

                vd.template getLastProp<vd9_volume>()[0] = dp;

                ++bound_box;
            }
        }
    }
}
int FindSector(real_number th, std::vector<double> angles)
{
    // angles contain angles defining circular sectors, in increasing order
    // need to find to which sector the angle th belongs to
    // return the index beginning from 0

    // normalzie th to [0, 2pi]
    while (th < 0)
        th += 2 * M_PI;
    while (th >= 2 * M_PI)
        th -= 2 * M_PI;
    // normalize angles to [0, 2pi]
    for (int i = 0; i < angles.size(); i++)
    {
        while (angles[i] < 0)
            angles[i] += 2 * M_PI;
        while (angles[i] >= 2 * M_PI)
            angles[i] -= 2 * M_PI;
    }

    int nsectors = angles.size();
    int sector = -1;
    // Handle the last sector wrapping around to the first
    for (int i = 0; i < nsectors; i++)
    {
        double start_angle = angles[i];
        double end_angle = (i + 1 < nsectors) ? angles[i + 1] : angles[0] + 2 * M_PI;

        if (th >= start_angle && th < end_angle)
        {
            sector = i;
            break;
        }
    }

    // printf("th = %f, sector = %d\n", th * 180.0 / M_PI, sector);

    return sector;
}
void CreateParticleGeometryFlower(particles &vd, std::vector<std::pair<probe_particles, int>> &vp_vec, Obstacle *&obstacle_ptr, Parameters &params, AuxiliarParameters &auxParams)
{

    Vcluster<> &v_cl = create_vcluster();

    // Non periodic situation grid of 5 fluid particles and 3 boundary particles
    // We need a virtual grid of 5 + 2*(3+1) particles,
    // therefore the domain is discretized with 13 grid points,
    // when we use DrawParticles::DrawBox we will draw only the particles at the grid positons strictly inside the box,
    // the () repesent the recipient box, and the || represent the fluid box, we can see how this distribution places exactly 5 fluid particles inside and 3 boundary particles
    //           D-(-o--o--o-|-x--x--x--x--x--|-o-o-o-)-D
    // D: domain, o: boundary, x: fluid, --: dp distance
    // in a periodic situation we have the following
    // .....--x--x--D-|-x--x--x--x--x--|-D--x--x--......
    // therefore we need a grid of 5 + 2 particles, and the domain is discretized with 7 grid points

    // Size of the virtual cartesian grid that defines where to place the particles
    size_t sz[DIM];

    obstacle_ptr = new EmptyObstacle(params);
    Obstacle *obstacle_ptr2 = new EmptyObstacle(params);

    real_number dp = params.dp;

    // obstacle_ptr = new CurveObstacle(params.flowerA, params.flowerB, params.flowerK, params.flowerM, params.ObstacleCenter, params, params.ObstacleVelocity, params.ObstacleOmega, params.rf);
    obstacle_ptr = new EpiCycloid(params.flowerPropRadius * params.flowerA, params.flowerNlobes, params.ObstacleCenter, params, params.ObstacleVelocity, 0.0, params.rf);
    obstacle_ptr2 = new HipoCycloid(params.flowerPropRadius * params.flowerA / params.flowerPropRadius2, params.flowerNlobes, params.ObstacleCenter, params, params.ObstacleVelocity, 0.0, params.rf);

    real_number rotor_pos_r = params.flowerPropRadius * (5.0 / 3.0) * params.flowerA / 2.0;

    std::vector<Obstacle *> rotors;
    int nlobes = (int)params.flowerNlobes;
    std::vector<real_number> ths;
    for (int kk = 0; kk < nlobes; kk++)
    {
        int sign = 1;
        if (params.flowerAlternate == 1)
            sign = std::pow(-1, kk);

        real_number th = M_PI / params.flowerNlobes + 2 * M_PI * kk / params.flowerNlobes;
        ths.push_back(2 * M_PI * kk / params.flowerNlobes);
        // printf("th = %f\n", 2 * M_PI * kk / params.flowerNlobes);
        Point<DIM, real_number> rotor_pos = params.ObstacleCenter;
        Point<DIM, real_number> offset = {rotor_pos_r * cos(th), rotor_pos_r * sin(th)};
        rotor_pos = rotor_pos + offset;
        Obstacle *tmp = new CurveObstacle(params.flowerA, params.flowerB, params.flowerK, params.flowerM, rotor_pos, params, params.ObstacleVelocity, sign * params.ObstacleOmega, params.rf);
        rotors.push_back(tmp);
    }

    // Now define the iterator boxes
    // We define the boxes in terms of offstes with respect to the fluid box that goes from 0 to length
    real_number offset_domain_left[DIM] = {0.0};
    real_number offset_domain_right[DIM] = {0.0};
    real_number offset_recipient[DIM] = {0.0};
    real_number offset_periodic_fluid[DIM] = {0.0};
    real_number offset_periodic_recipient[DIM] = {0.0};

    for (int xyz = 0; xyz < DIM; xyz++)
    {
        if (params.bc[xyz] == NON_PERIODIC) // non periodic, fluid covered by boundary
        {
            sz[xyz] = params.Nfluid[xyz] + 2 * (params.Nboundary[xyz] + 1);
            offset_domain_left[xyz] = (0.5 + params.Nboundary[xyz]) * dp;
            offset_domain_right[xyz] = (0.5 + params.Nboundary[xyz]) * dp;

            if (params.BC_TYPE == NEW_NO_SLIP) // Nboundary should only be 0 or 1 if we are using the new bc
                offset_recipient[xyz] = 0.1 * params.Nboundary[xyz] * dp;
            else if (params.BC_TYPE == NO_SLIP)
                offset_recipient[xyz] = params.Nboundary[xyz] * dp;
        }
        else // periodic, open ended
        {
            sz[xyz] = params.Nfluid[xyz] + 1;

            offset_domain_left[xyz] = 0.0;
            offset_domain_right[xyz] = dp;
            offset_periodic_fluid[xyz] = 0.75 * dp;
            offset_periodic_recipient[xyz] = 0.85 * dp;
        }
    }

    // Define the boxes
    Box<DIM, real_number> domain({-offset_domain_left[0],
                                  -offset_domain_left[1]},
                                 {params.length[0] + offset_domain_right[0],
                                  params.length[1] + offset_domain_right[1]});

    Box<DIM, real_number> fluid_box({0.0,
                                     0.0},
                                    {params.length[0] + offset_periodic_fluid[0],
                                     params.length[1] + offset_periodic_fluid[1]});

    Box<DIM, real_number> recipient({-offset_recipient[0],
                                     -offset_recipient[1]},
                                    {params.length[0] + offset_recipient[0] + offset_periodic_recipient[0],
                                     params.length[1] + offset_recipient[1] + offset_periodic_recipient[1]});

    // Will only be used in the new bc
    // Box<DIM, real_number> recipient_hole({offset_recipient[0],
    //                                       offset_recipient[1]},
    //                                      {params.length[0] - offset_recipient[0] + offset_periodic_fluid[0],
    //                                       params.length[1] - offset_recipient[1] + offset_periodic_fluid[1]});

    for (int xyz = 0; xyz < DIM; xyz++) // correct length in periodic case
    {
        if (params.bc[xyz] == PERIODIC)
            params.length[xyz] += dp;
    }

    // extended boundary around the domain, and the processor domain
    Ghost<DIM, real_number> g(params.r_cut);

    // create particle object
    particles vd_loc(0, domain, params.bc, g, DEC_GRAN(128));
    // vd is argument passed as reference we want to fill with particles
    vd = vd_loc;

    // For the new bc we place the wall/obstacle particles manually
    // Add the obstacle/walls as marker particles only on processor 0
    if (params.BC_TYPE == NEW_NO_SLIP && v_cl.getProcessUnitID() == 0)
    {
        // Add obstacle
        obstacle_ptr->AddObstacle(vd);
        obstacle_ptr2->AddObstacle(vd);

        for (auto &rotor : rotors)
        {
            rotor->AddObstacle(vd);
        }
    }

    // return an iterator to the fluid particles to add to vd
    auto fluid_it = DrawParticles::DrawBox(vd, sz, domain, fluid_box);

    // for each particle inside the fluid box ...
    if (params.BC_TYPE == NEW_NO_SLIP)
    {
        while (fluid_it.isNext())
        {

            Point<DIM, real_number> iterator_position = fluid_it.get();

            if (IsInsideAll(rotors, iterator_position) || !(*obstacle_ptr).isInside(iterator_position) || (*obstacle_ptr2).isInside(iterator_position)) // if not inside the obstacles, or outside the containing obstacle
            {
                ++fluid_it;
                continue;
            }
            else // if not inside obstacle at all just add fluid particles
            {
                // ... add a particle ...
                vd.add();
                vd.template getLastProp<vd0_type>() = FLUID;
                vd.template getLastProp<vd10_omega>() = 0.0;
                for (int xyz = 0; xyz < DIM; xyz++)
                {
                    vd.template getLastProp<vd4_velocity>()[xyz] = 0.0;
                    vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
                }
            }

            // Set properties common to fluid and obstacle particles
            vd.template getLastProp<vd1_rho>() = params.rho0;
            vd.template getLastProp<vd2_pressure>() = 0.0;
            vd.template getLastProp<vd3_drho>() = 0.0;

            for (int xyz = 0; xyz < DIM; xyz++)
            {
                vd.getLastPos()[xyz] = iterator_position.get(xyz);
                vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
                vd.template getLastProp<vd8_normal>()[xyz] = 0.0;
                vd.template getLastProp<vd6_force>()[xyz] = 0.0;
            }

            // fluid particles carry dp in volume[0], it is useful to read it in postprocessing
            // this array is unused for fluid particles anyway
            vd.template getLastProp<vd9_volume>()[0] = dp;

            real_number theta = atan2(iterator_position.get(1) - params.ObstacleCenter[1], iterator_position.get(0) - params.ObstacleCenter[0]);

            int idx = FindSector(theta, ths);

            // if ((iterator_position.get(0) > params.ObstacleCenter[0]) && (iterator_position.get(1) > params.ObstacleCenter[1]))
            //     vd.template getLastProp<vd9_volume>()[1] = 100.0;
            // else if ((iterator_position.get(0) < params.ObstacleCenter[0]) && (iterator_position.get(1) > params.ObstacleCenter[1]))
            //     vd.template getLastProp<vd9_volume>()[1] = 200.0;
            // else if ((iterator_position.get(0) < params.ObstacleCenter[0]) && (iterator_position.get(1) < params.ObstacleCenter[1]))
            //     vd.template getLastProp<vd9_volume>()[1] = 300.0;
            // else if ((iterator_position.get(0) > params.ObstacleCenter[0]) && (iterator_position.get(1) < params.ObstacleCenter[1]))
            //     vd.template getLastProp<vd9_volume>()[1] = 400.0;
            // else
            //     vd.template getLastProp<vd9_volume>()[1] = 0.0;

            vd.template getLastProp<vd9_volume>()[1] = 100.0 * idx;

            vd.template getLastProp<vd9_volume>()[2] = 0.0;

            vd.template getLastProp<vd11_vorticity>() = 0.0;
            vd.template getLastProp<vd12_vel_red>() = 0.0;
            vd.template getLastProp<vd13_force_red_x>() = 0.0;
            vd.template getLastProp<vd14_force_red_y>() = 0.0;

            // next fluid particle
            ++fluid_it;
        }
    }
    else
    {
        while (fluid_it.isNext())
        {

            Point<DIM, real_number> iterator_position = fluid_it.get();

            if (IsInsideAll(rotors, iterator_position) || !(*obstacle_ptr).isInside(iterator_position) || (*obstacle_ptr2).isInside(iterator_position)) // if not inside the obstacles, or outside the containing obstacle
            {
                // ... add a particle ...
                vd.add();
                vd.template getLastProp<vd0_type>() = OBSTACLE;
                vd.template getLastProp<vd10_omega>() = 0.0;

                for (int xyz = 0; xyz < DIM; xyz++)
                {
                    vd.template getLastProp<vd4_velocity>()[xyz] = 0.0;
                    vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
                }

                for (auto rotor : rotors)
                {
                    if (rotor->isInside(iterator_position))
                    {
                        vd.template getLastProp<vd10_omega>() = rotor->AngularVelocity_;
                        for (int xyz = 0; xyz < DIM; xyz++)
                        {
                            vd.template getLastProp<vd4_velocity>()[xyz] = (rotor->LinearVelocity_).get(xyz);
                            vd.template getLastProp<vd7_force_t>()[xyz] = (rotor->Centre_).get(xyz);
                        }
                    }
                }
            }
            else // if not inside obstacle at all just add fluid particles
            {
                // ... add a particle ...
                vd.add();
                vd.template getLastProp<vd0_type>() = FLUID;
                vd.template getLastProp<vd10_omega>() = 0.0;
                for (int xyz = 0; xyz < DIM; xyz++)
                {
                    vd.template getLastProp<vd4_velocity>()[xyz] = 0.0;
                    vd.template getLastProp<vd7_force_t>()[xyz] = 0.0;
                }
            }

            // Set properties common to fluid and obstacle particles
            vd.template getLastProp<vd1_rho>() = params.rho0;
            vd.template getLastProp<vd2_pressure>() = 0.0;
            vd.template getLastProp<vd3_drho>() = 0.0;

            for (int xyz = 0; xyz < DIM; xyz++)
            {
                vd.getLastPos()[xyz] = iterator_position.get(xyz);
                vd.template getLastProp<vd5_velocity_t>()[xyz] = 0.0;
                vd.template getLastProp<vd8_normal>()[xyz] = 0.0;
                vd.template getLastProp<vd6_force>()[xyz] = 0.0;
            }

            // fluid particles carry dp in volume[0], it is useful to read it in postprocessing
            // this array is unused for fluid particles anyway
            vd.template getLastProp<vd9_volume>()[0] = dp;

            real_number theta = atan2(iterator_position.get(1) - params.ObstacleCenter[1], iterator_position.get(0) - params.ObstacleCenter[0]);

            int idx = FindSector(theta, ths);

            // if ((iterator_position.get(0) > params.ObstacleCenter[0]) && (iterator_position.get(1) > params.ObstacleCenter[1]))
            //     vd.template getLastProp<vd9_volume>()[1] = 100.0;
            // else if ((iterator_position.get(0) < params.ObstacleCenter[0]) && (iterator_position.get(1) > params.ObstacleCenter[1]))
            //     vd.template getLastProp<vd9_volume>()[1] = 200.0;
            // else if ((iterator_position.get(0) < params.ObstacleCenter[0]) && (iterator_position.get(1) < params.ObstacleCenter[1]))
            //     vd.template getLastProp<vd9_volume>()[1] = 300.0;
            // else if ((iterator_position.get(0) > params.ObstacleCenter[0]) && (iterator_position.get(1) < params.ObstacleCenter[1]))
            //     vd.template getLastProp<vd9_volume>()[1] = 400.0;
            // else
            //     vd.template getLastProp<vd9_volume>()[1] = 0.0;

            if (vd.template getLastProp<vd0_type>() == FLUID)
                vd.template getLastProp<vd9_volume>()[1] = 100.0 * idx;

            vd.template getLastProp<vd9_volume>()[2] = 0.0;

            vd.template getLastProp<vd11_vorticity>() = 0.0;
            vd.template getLastProp<vd12_vel_red>() = 0.0;
            vd.template getLastProp<vd13_force_red_x>() = 0.0;
            vd.template getLastProp<vd14_force_red_y>() = 0.0;

            // next fluid particle
            ++fluid_it;
        }
    }
}
